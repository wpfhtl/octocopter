#include "hip/hip_runtime.h"
// Fix for gcc 4.7
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include "thrust/device_ptr.h"
#include "thrust/device_vector.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particleskernel.cu"
#include "hip/hip_runtime.h"

#include <QDebug>

void copyParametersToGpu(SimulationParameters *hostParams)
{
    // Copy parameters to constant memory. This was synchronous once, I changed
    // it to be asynchronous. Shouldn't cause any harm, even if parameters were
    // applied one frame too late.
    hipMemcpyToSymbol/*Async*/(params, hostParams, sizeof(SimulationParameters));
}

void integrateSystem(float *particlePositions, float *particleVelocities, uint8_t* gridWaypointPressure, float* particleCollisionPositions, float deltaTime, uint numParticles)
{
// old thrust version. Cannot write to the non-linear waypointpressure position when using thrust tuples.
//    thrust::device_ptr<float4> d_pos4((float4*)pos);
//    thrust::device_ptr<float4> d_vel4((float4*)vel);
//    thrust::device_ptr<float4> d_pcp4((float4*)particleCollisionPositions);
//    thrust::device_ptr<uint8_t> d_gwpp((uint8_t*)gridWaypointPressure);

//    thrust::for_each(
//                thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4, d_pcp4, d_gwpp)),
//                thrust::make_zip_iterator(thrust::make_tuple(d_pos4 + numParticles, d_vel4 + numParticles, d_pcp4 + numParticles, d_gwpp + numParticles)),
//                integrate_functor(deltaTime));

    if(numParticles == 0) return;

    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    integrateSystemD<<< numBlocks, numThreads >>>(
                                                    (float4*)particlePositions,          // in/out: particle positions
                                                    (float4*)particleVelocities,         // in/out: particle velocities
                                                    gridWaypointPressure,       // in/out: grid containing quint8-cells with waypoint-pressure values (80-255)
                                                    (float4*)particleCollisionPositions, // input:  particle positions
                                                    deltaTime,
                                                    numParticles);

    // check if kernel invocation generated an error
    cudaCheckSuccess("integrateSystem");
}

// Calculates a hash for each particle. The hash value is ("based on") its cell id.
void computeMappingFromGridCellToParticle(
        uint*  gridParticleHash,
        uint*  gridParticleIndex,
        float* pos,
        int    numParticles)
{
    if(numParticles == 0) return;

    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    computeMappingFromGridCellToParticleD<<< numBlocks, numThreads >>>(gridParticleHash,
                                           gridParticleIndex,
                                           (float4 *) pos,
                                           numParticles);

    // check if kernel invocation generated an error
    cudaCheckSuccess("computeMappingFromGridCellToParticleD");
}

void sortParticlePosAndVelAccordingToGridCellAndFillCellStartAndEndArrays(
        uint*  cellStart,
        uint*  cellEnd,
        float* sortedPos,
        float* sortedVel,
        uint*  gridParticleHash,
        uint*  gridParticleIndex,
        float* oldPos,
        float* oldVel,
        uint   numParticles,
        uint   numCells)
{
    // set all cells to empty
    hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint));

    if(numParticles == 0) return;

    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numParticles, 256, numBlocks, numThreads);


#if USE_TEX
    hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4));
    if(oldVel && sortedVel) hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4));
#endif

    // Number of bytes in shared memory that is allocated for each (thread)block.
    uint smemSize = sizeof(uint)*(numThreads+1);

    sortPosAndVelAccordingToGridCellAndFillCellStartAndEndArraysD<<< numBlocks, numThreads, smemSize>>>(
                                                                         cellStart,
                                                                         cellEnd,
                                                                         (float4 *) sortedPos,
                                                                         (float4 *) sortedVel,
                                                                         gridParticleHash,
                                                                         gridParticleIndex,
                                                                         (float4 *) oldPos,
                                                                         (float4 *) oldVel,
                                                                         numParticles);

    cudaCheckSuccess("sortPosAndVelAccordingToGridCellAndFillCellStartAndEndArraysD");

#if USE_TEX
    hipUnbindTexture(oldPosTex);
    if(oldVel && sortedVel) hipUnbindTexture(oldVelTex);
#endif
}

void collideParticlesWithParticlesAndColliders(
        float* newVel,              // output: The particle velocities
        float* particleCollisionPositions,          // output: Every particle's position of last collision, or 0.0/0.0/0.0 if none occurred.

        float* particlePosSorted,   // input:  The particle positions, sorted by gridcell
        float* particleVelSorted,   // input:  The particle velocities, sorted by gridcell
        uint*  particleMapIndex,    // input:  The value-part of the particle gridcell->index map, sorted by gridcell
        uint*  particleCellStart,   // input:  At which index in mDeviceMapParticleIndex does cell X start?
        uint*  particleCellEnd,     // input:  At which index in mDeviceMapParticleIndex does cell X end?

        float* colliderSortedPos,   // input:  The collider positions, sorted by gridcell
        uint*  colliderMapIndex,    // input:  The value-part of the collider gridcell->index map, sorted by gridcell
        uint*  colliderCellStart,   // input:  At which index in mDeviceMapColliderIndex does cell X start?
        uint*  colliderCellEnd,     // input:  At which index in mDeviceMapColliderIndex does cell X end?

        uint   numParticles,        // input:  How many particles to collide against other particles (one thread per particle)
        uint   numCells             // input:  Number of grid cells
        )
{
#if USE_TEX
    hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4));
    hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4));
    hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint));
    hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint));
#endif

    // thread per particle
    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numParticles, 64, numBlocks, numThreads);

    // execute the kernel
    collideParticlesWithParticlesAndCollidersD<<< numBlocks, numThreads >>>(
                                                                              (float4*)newVel,
                                                                              (float4*)particleCollisionPositions,

                                                                              (float4*)particlePosSorted,
                                                                              (float4*)particleVelSorted,
                                                                              particleMapIndex,
                                                                              particleCellStart,
                                                                              particleCellEnd,

                                                                              (float4*)colliderSortedPos,
                                                                              colliderMapIndex,
                                                                              colliderCellStart,
                                                                              colliderCellEnd,

                                                                              numParticles);

    // check if kernel invocation generated an error
    cudaCheckSuccess("collideParticlesWithParticlesD");

#if USE_TEX
    hipUnbindTexture(oldPosTex);
    hipUnbindTexture(oldVelTex);
    hipUnbindTexture(cellStartTex);
    hipUnbindTexture(cellEndTex);
#endif
}

void sortGridOccupancyMap(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
{
    if(numParticles > 0)
        thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),                // KeysBeginning
                            thrust::device_ptr<uint>(dGridParticleHash + numParticles), // KeysEnd
                            thrust::device_ptr<uint>(dGridParticleIndex));              // ValuesBeginning

    // check if kernel invocation generated an error
    cudaCheckSuccess("sortGridOccupancyMap");
}


// Fill mDeviceGridMapCellWorldPositions - this might be done only once and then copied lateron (just like the waypoint pressure above)
void fillGridMapCellWorldPositions(float* gridMapCellWorldPositions, uint numCells)
{
    // thread per cell
    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numCells, 64, numBlocks, numThreads);

    fillGridMapCellWorldPositionsD<<< numBlocks, numThreads >>>(
                                                                  (float4*)gridMapCellWorldPositions,
                                                                  numCells);
}

void moveGridMapWayPointPressureValuesByWorldPositionOffset(quint8 *gridMapOfWayPointPressure, float* offset, unsigned int numberOfCells)
{
    // thread per cell
    uint numThreads, numBlocks;
    computeExecutionKernelGrid(1/*numberOfCells*/, 64, numBlocks, numThreads);

    qDebug() << "kernel exec:" << numberOfCells << numBlocks << numThreads << offset[0] << offset[1] << offset[2];

    moveGridMapWayPointPressureValuesByWorldPositionOffsetD<<< numBlocks, numThreads >>>(
                                                                                           (uint8_t*)gridMapOfWayPointPressure,
                                                                                           (float3*)offset,
                                                                                           numberOfCells);
    qDebug() << "kernel exec done, checking...";

    cudaCheckSuccess("moveGridMapWayPointPressureValuesByWorldPositionOffset");
}

// Sort mDeviceGridMapWayPointPressureSorted => mDeviceGridMapCellWorldPositions according to the keys DESC
void sortGridMapWayPointPressure(uint8_t* gridMapWayPointPressureSorted, float* gridMapCellWorldPositions, uint numCells, uint numWaypointsRequested)
{
    if(numCells > 0)
    {
        thrust::sort_by_key(thrust::device_ptr<uint8_t>(gridMapWayPointPressureSorted),             // KeysBeginning
                            thrust::device_ptr<uint8_t>(gridMapWayPointPressureSorted + numCells),  // KeysEnd
                            thrust::device_ptr<float4>((float4*)gridMapCellWorldPositions),         // ValuesBeginning
                            thrust::greater<int>());                                                // In descending order

        // Now we want to copy the waypointpressure-value for all requested waypoints from gridMapWayPointPressureSorted(quint8) to gridMapCellWorldPositions.w(float)
        thrust::device_ptr<float4>  d_cwp((float4*)gridMapCellWorldPositions);
        thrust::device_ptr<uint8_t> d_wpp((uint8_t*)gridMapWayPointPressureSorted);

        thrust::for_each(
                    thrust::make_zip_iterator(thrust::make_tuple(d_wpp, d_cwp)),
                    thrust::make_zip_iterator(thrust::make_tuple(d_wpp + numWaypointsRequested, d_cwp + numWaypointsRequested)),
                    copy_functor());
    }

    // check if kernel invocation generated an error
    cudaCheckSuccess("sortGridMapWayPointPressure");
}
