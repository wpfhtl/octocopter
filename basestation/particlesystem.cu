#include "hip/hip_runtime.h"
// Fix for gcc 4.7
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include "thrust/device_ptr.h"
#include "thrust/device_vector.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particleskernel.cu"
#include "hip/hip_runtime.h"

#include <QDebug>

// was 64
#define KERNEL_LAUNCH_BLOCKSIZE 256

void copyParametersToGpu(SimulationParameters *hostParams)
{
    // Copy parameters to constant memory. This was synchronous once, I changed
    // it to be asynchronous. Shouldn't cause any harm, even if parameters were
    // applied one frame too late.
    hipMemcpyToSymbol/*Async*/(params, hostParams, sizeof(SimulationParameters));
}

void integrateSystem(float *particlePositions, float *particleVelocities, uint8_t* gridWaypointPressure, float* particleCollisionPositions, uint numParticles)
{
// old thrust version. Cannot write to the non-linear waypointpressure position when using thrust tuples.
//    thrust::device_ptr<float4> d_pos4((float4*)pos);
//    thrust::device_ptr<float4> d_vel4((float4*)vel);
//    thrust::device_ptr<float4> d_pcp4((float4*)particleCollisionPositions);
//    thrust::device_ptr<uint8_t> d_gwpp((uint8_t*)gridWaypointPressure);

//    thrust::for_each(
//                thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4, d_pcp4, d_gwpp)),
//                thrust::make_zip_iterator(thrust::make_tuple(d_pos4 + numParticles, d_vel4 + numParticles, d_pcp4 + numParticles, d_gwpp + numParticles)),
//                integrate_functor(deltaTime));

    if(numParticles == 0) return;

    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numParticles, KERNEL_LAUNCH_BLOCKSIZE, numBlocks, numThreads);

    // execute the kernel
    integrateSystemD<<< numBlocks, numThreads >>>(
                                                    (float4*)particlePositions,          // in/out: particle positions
                                                    (float4*)particleVelocities,         // in/out: particle velocities
                                                    gridWaypointPressure,       // in/out: grid containing quint8-cells with waypoint-pressure values (80-255)
                                                    (float4*)particleCollisionPositions, // input:  particle positions
                                                    numParticles);

    // check if kernel invocation generated an error
    cudaCheckSuccess("integrateSystem");
}

// Calculates a hash for each particle. The hash value is ("based on") its cell id.
void computeMappingFromGridCellToParticle(
        uint*  gridParticleHash,
        uint*  gridParticleIndex,
        float* pos,
        int    numParticles)
{
    if(numParticles == 0) return;

    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numParticles, KERNEL_LAUNCH_BLOCKSIZE, numBlocks, numThreads);

    // execute the kernel
    computeMappingFromGridCellToParticleD<<< numBlocks, numThreads >>>(gridParticleHash,
                                           gridParticleIndex,
                                           (float4 *) pos,
                                           numParticles);

    // check if kernel invocation generated an error
    cudaCheckSuccess("computeMappingFromGridCellToParticleD");
}

void sortParticlePosAndVelAccordingToGridCellAndFillCellStartAndEndArrays(
        uint*  cellStart,
        uint*  cellEnd,
        float* sortedPos,
        float* sortedVel,
        uint*  gridParticleHash,
        uint*  gridParticleIndex,
        float* oldPos,
        float* oldVel,
        uint   numParticles,
        uint   numCells)
{
    // set all cells to empty
    hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint));

    if(numParticles == 0) return;

    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numParticles, KERNEL_LAUNCH_BLOCKSIZE, numBlocks, numThreads);


#if USE_TEX
    hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4));
    if(oldVel && sortedVel) hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4));
#endif

    // Number of bytes in shared memory that is allocated for each (thread)block.
    uint smemSize = sizeof(uint)*(numThreads+1);

    sortPosAndVelAccordingToGridCellAndFillCellStartAndEndArraysD<<< numBlocks, numThreads, smemSize>>>(
                                                                         cellStart,
                                                                         cellEnd,
                                                                         (float4 *) sortedPos,
                                                                         (float4 *) sortedVel,
                                                                         gridParticleHash,
                                                                         gridParticleIndex,
                                                                         (float4 *) oldPos,
                                                                         (float4 *) oldVel,
                                                                         numParticles);

    cudaCheckSuccess("sortPosAndVelAccordingToGridCellAndFillCellStartAndEndArraysD");

#if USE_TEX
    hipUnbindTexture(oldPosTex);
    if(oldVel && sortedVel) hipUnbindTexture(oldVelTex);
#endif
}

void collideParticlesWithParticlesAndColliders(
        float* newVel,              // output: The particle velocities
        float *particlePosVbo,      // output: The w-component is changed whenever a particle has hit a collider. Used just for visualization.
        float* particleCollisionPositions,          // output: Every particle's position of last collision, or 0.0/0.0/0.0 if none occurred.

        float* particlePosSorted,   // input:  The particle positions, sorted by gridcell
        float* particleVelSorted,   // input:  The particle velocities, sorted by gridcell
        uint*  particleMapIndex,    // input:  The value-part of the particle gridcell->index map, sorted by gridcell
        uint*  particleCellStart,   // input:  At which index in mDeviceMapParticleIndex does cell X start?
        uint*  particleCellEnd,     // input:  At which index in mDeviceMapParticleIndex does cell X end?

        float* colliderSortedPos,   // input:  The collider positions, sorted by gridcell
        uint*  colliderMapIndex,    // input:  The value-part of the collider gridcell->index map, sorted by gridcell
        uint*  colliderCellStart,   // input:  At which index in mDeviceMapColliderIndex does cell X start?
        uint*  colliderCellEnd,     // input:  At which index in mDeviceMapColliderIndex does cell X end?

        uint   numParticles,        // input:  How many particles to collide against other particles (one thread per particle)
        uint   numCells             // input:  Number of grid cells
        )
{

    // thread per particle
    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numParticles, KERNEL_LAUNCH_BLOCKSIZE, numBlocks, numThreads);

    // execute the kernel
    collideParticlesWithParticlesAndCollidersD<<< numBlocks, numThreads >>>(
                                                                              (float4*)newVel,
                                                                              (float4*)particlePosVbo,
                                                                              (float4*)particleCollisionPositions,

                                                                              (float4*)particlePosSorted,
                                                                              (float4*)particleVelSorted,
                                                                              particleMapIndex,
                                                                              particleCellStart,
                                                                              particleCellEnd,

                                                                              (float4*)colliderSortedPos,
                                                                              colliderMapIndex,
                                                                              colliderCellStart,
                                                                              colliderCellEnd,

                                                                              numParticles);

    // check if kernel invocation generated an error
    cudaCheckSuccess("collideParticlesWithParticlesD");
}

void sortGridOccupancyMap(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
{
    if(numParticles > 0)
        thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),                // KeysBeginning
                            thrust::device_ptr<uint>(dGridParticleHash + numParticles), // KeysEnd
                            thrust::device_ptr<uint>(dGridParticleIndex));              // ValuesBeginning

    // check if kernel invocation generated an error
    cudaCheckSuccess("sortGridOccupancyMap");
}


// Fill mDeviceGridMapCellWorldPositions - this might be done only once and then copied lateron (just like the waypoint pressure above)
void fillGridMapCellWorldPositions(float* gridMapCellWorldPositions, uint numCells)
{
    // thread per cell
    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numCells, KERNEL_LAUNCH_BLOCKSIZE, numBlocks, numThreads);

    fillGridMapCellWorldPositionsD<<< numBlocks, numThreads >>>(
                                                                  (float4*)gridMapCellWorldPositions,
                                                                  numCells);
}

// Sort mDeviceGridMapWayPointPressureSorted => mDeviceGridMapCellWorldPositions according to the keys DESC
void sortGridMapWayPointPressure(float* gridMapWayPointPressureSorted, float* gridMapCellWorldPositions, uint numberOfCells, uint numWaypointsRequested)
{
    if(numberOfCells > 0)
    {
        thrust::sort_by_key(thrust::device_ptr<float>(gridMapWayPointPressureSorted),             // KeysBeginning
                            thrust::device_ptr<float>(gridMapWayPointPressureSorted + numberOfCells),  // KeysEnd
                            thrust::device_ptr<float4>((float4*)gridMapCellWorldPositions),         // ValuesBeginning
                            thrust::greater<float>());                                                // In descending order

        // Now we want to copy the waypointpressure-value for all requested waypoints from gridMapWayPointPressureSorted(quint8) to gridMapCellWorldPositions.w(float)
        thrust::device_ptr<float4>  d_cwp((float4*)gridMapCellWorldPositions);
        thrust::device_ptr<float> d_wpp((float*)gridMapWayPointPressureSorted);

        thrust::for_each(
                    thrust::make_zip_iterator(thrust::make_tuple(d_wpp, d_cwp)),
                    thrust::make_zip_iterator(thrust::make_tuple(d_wpp + numWaypointsRequested, d_cwp + numWaypointsRequested)),
                    copy_functor());
    }

    // check if kernel invocation generated an error
    cudaCheckSuccess("sortGridMapWayPointPressure");
}

uint8_t getMaximumWaypointPressure(uint8_t* gridMapOfWayPointPressure, unsigned int numberOfCells)
{
    if(numberOfCells > 0)
    {
        thrust::device_ptr<uint8_t> result = thrust::max_element(
                    thrust::device_ptr<uint8_t>(gridMapOfWayPointPressure),
                    thrust::device_ptr<uint8_t>(gridMapOfWayPointPressure + numberOfCells));

        // check if kernel invocation generated an error
        cudaCheckSuccess("getMaximumWaypointPressure");

        return *result;
    }

    return 0;
}

void decreaseWaypointPressure(uint8_t* gridMapOfWayPointPressure, unsigned int numberOfCells)
{
    if(numberOfCells > 0)
    {

        // in-place transformation
        thrust::transform(thrust::device_ptr<uint8_t>(gridMapOfWayPointPressure),
                          thrust::device_ptr<uint8_t>(gridMapOfWayPointPressure + numberOfCells),
                          thrust::device_ptr<uint8_t>(gridMapOfWayPointPressure),
                          functorDecreaseWaypointPressure());

    }

    // check if kernel invocation generated an error
    cudaCheckSuccess("decreaseWaypointPressure");
}

void computeWaypointBenefit(float* gridMapOfWayPointPressureDst, uint8_t* gridMapOfWayPointPressureSrc, float* vehiclePosition, unsigned int numberOfCells)
{
    if(numberOfCells > 0)
    {
        //    thrust::device_ptr<float4> d_pos4((float4*)pos);
        //    thrust::device_ptr<float4> d_vel4((float4*)vel);
        //    thrust::device_ptr<float4> d_pcp4((float4*)particleCollisionPositions);
        //    thrust::device_ptr<uint8_t> d_gwpp((uint8_t*)gridWaypointPressure);


        thrust::counting_iterator<unsigned int> cellHash(0);
        thrust::device_ptr<uint8_t> d_wpp_src(gridMapOfWayPointPressureSrc);
        thrust::device_ptr<float> d_wpp_dst(gridMapOfWayPointPressureDst);

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_wpp_src, d_wpp_dst, cellHash)),
            thrust::make_zip_iterator(thrust::make_tuple(d_wpp_src + numberOfCells, d_wpp_dst + numberOfCells, cellHash + numberOfCells)),
            functorComputeWaypointBenefit(*((float3*)vehiclePosition)));

        /*
        // in-place transformation
        thrust::transform(thrust::device_ptr<uint8_t>(gridMapOfWayPointPressureSrc),
                          thrust::device_ptr<uint8_t>(gridMapOfWayPointPressureSrc + numberOfCells),
                          thrust::device_ptr<uint8_t>(gridMapOfWayPointPressureDst),
                          functorComputeWaypointBenefit((float3*)vehiclePosition));
        */

    }

    // check if kernel invocation generated an error
    cudaCheckSuccess("computeWaypointBenefit");
}
