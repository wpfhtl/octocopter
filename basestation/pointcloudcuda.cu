#include "hip/hip_runtime.h"
// Fix for gcc 4.7
//#undef _GLIBCXX_ATOMIC_BUILTINS
//#undef _GLIBCXX_USE_INT128

#include "thrust/sort.h"
#include "thrust/unique.h"
#include <thrust/remove.h>
#include <thrust/count.h>
#include <thrust/device_ptr.h>

#include "hip/hip_runtime.h"
#include "cudahelper.cuh"
#include "pointcloudcuda.cuh"
#include "hip/hip_vector_types.h"
#include "grid.cuh"

// pointcloud parameters in constant memory
__constant__ ParametersPointCloud paramsPointCloud;

inline __host__ __device__ bool operator!=(float3 &a, float3 &b)
{
    return !(a.x == b.x && a.y == b.y && a.z == b.z);
}

inline __host__ __device__ bool operator!=(float4 &a, float4 &b)
{
    return !(a.x == b.x && a.y == b.y && a.z == b.z && a.w == b.w);
}

// rearrange particle data into sorted order (sorted according to containing grid cell), and find the start of each cell in the sorted hash array
void getDeviceAddressOfParametersPointCloud(ParametersPointCloud** ptr)
{
    cudaSafeCall(hipGetSymbolAddress((void**)ptr, paramsPointCloud));
}

void copyParametersToGpu(ParametersPointCloud *hostParams)
{
    // copy parameters to constant memory
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(paramsPointCloud), hostParams, sizeof(ParametersPointCloud)));
}

// Returns the presence of neighbors of @pos within paramsPointCloud.minimumDistance in @gridCell
// Caller must ensure:
//  - pos is in grid
//  - gridCell is valid (less than grid.cells)
__device__ void checkCellForNeighbors(
        int3    gridCell,       // grid cell to search for particles that could collide
        uint    index,          // index of particle that is being collided
        float4  pos,            // position of particle that is being collided
        float4* posSorted,
        uint*   pointCellStart,
        uint*   pointCellStopp,
        float4& clusterPosition,
        uint&   numberofNeighborsFound)
{
    const uint gridHash = paramsPointCloud.grid.getCellHash(gridCell);

    // get start of bucket for this cell
    uint startIndex = pointCellStart[gridHash];

    // cell is not empty
    if(startIndex != 0xffffffff)
    {
        // iterate over particles in this cell
        uint endIndex = pointCellStopp[gridHash];

        for(uint j=startIndex; j<endIndex; j++)
        {
            // check not colliding with self
            if(j != index)
            {
                const float4 posOther = posSorted[j];
                const float4 relPos = pos - posOther;
                float distSquared = lengthSquared(make_float3(relPos));

                // If they collide AND we're checking the point that was further from the scanner, THEN reduce it!
                if(distSquared < paramsPointCloud.minimumDistance * paramsPointCloud.minimumDistance)
                {
                    clusterPosition += posOther;
                    numberofNeighborsFound ++;
                }
            }
        }
    }
}

// not really random, but should be good enough.
__device__ float randomNumber(uint seed)
{
    uint a = threadIdx.x * seed;
    uint b = blockIdx.x * blockDim.x;

    b = 36969 * (b & 65535) + (b >> 16);
    a = 18000 * (a & 65535) + (a >> 16);

    return ((b << 16) + a) / 4294967295.0;
}

// Collide a single point (given by thread-id through @index) against all points in own and neighboring cells
__global__
void markCollidingPointsD(
        float4* posOriginal,        // output: new positions, same or zeroed. This is actually mDevicePointPos, so its the original position location
        float4* positionsSorted,    // input: positions sorted according to containing grid cell
        uint*   gridPointIndex,     // input: particle indices sorted according to containing grid cell
        uint*   pointCellStart,     // input: pointCellStart[19] contains the index of gridParticleIndex in which cell 19 starts
        uint*   pointCellStopp,     // input: pointCellStopp[19] contains the index of gridParticleIndex in which cell 19 ends
        uint    numPoints)          // input: number of total particles
{
    uint threadIndex = getThreadIndex1D();
    if(threadIndex >= numPoints) return;

    // read particle data from sorted arrays
    const float4 worldPos = positionsSorted[threadIndex];

    // get address of particle in grid
    const int3 gridCellCoordinate = paramsPointCloud.grid.getCellCoordinate(make_float3(worldPos));

    // Do not process points that are not in the defined grid!
    if(gridCellCoordinate.x == -1)
    {
        printf("got a point not in grid, ouch!\n");
        return;
    }

    const uint originalIndex = gridPointIndex[threadIndex];

    float4 clusterPosition = make_float4(0.0);
    unsigned int numberOfCollisionsInOwnCell = 0;
    unsigned int numberOfCollisionsInNeighborCells = 0;

    // This code tries to optimize, thinking: If we already find many neighbors in our own cell, there's really not much
    // use in looking in other cells, too. We could even go so far as to test for the minimumDistance vs cellSize: If the
    // mindist is 20 cm, the cell contains 100 points and the cellsize is 20cm, then we know we have lots of neighbors
    // without even looking at them. But we don't know their .w component values!
    checkCellForNeighbors(
                gridCellCoordinate,
                threadIndex,
                worldPos,
                positionsSorted,
                pointCellStart,
                pointCellStopp,
                clusterPosition,
                numberOfCollisionsInOwnCell);

    // examine neighbouring cells
    for(int z=-1; z<=1 && numberOfCollisionsInOwnCell < 5; z++)
    {
        for(int y=-1; y<=1; y++)
        {
            for(int x=-1; x<=1; x++)
            {
                const int3 neighbourGridCoordinate = gridCellCoordinate + make_int3(x, y, z);
                if(x == 0 && y == 0 && z == 0) continue;

                checkCellForNeighbors(
                            neighbourGridCoordinate,
                            threadIndex,
                            worldPos,
                            positionsSorted,
                            pointCellStart,
                            pointCellStopp,
                            clusterPosition,
                            numberOfCollisionsInNeighborCells);
            }
        }
    }

    const float numberOfCollisionsTotal = numberOfCollisionsInOwnCell + numberOfCollisionsInNeighborCells;
    if(numberOfCollisionsTotal > 0.0)
    {
        const float averageNeighborScanDistance = clusterPosition.w / numberOfCollisionsTotal;
        if(averageNeighborScanDistance > worldPos.w /*&& randomNumber(numPoints + threadIndex) * numberOfCollisionsTotal > 1.0*/)
        {
            // If the other neighbors are of better quality, delete ourselves
            posOriginal[originalIndex] = make_float4(0.0);
        }
        else
        {
            // If we're better, move us into the center of our neighborhood
            posOriginal[originalIndex] = clusterPosition / numberOfCollisionsTotal;
        }
    }

    //posOriginal[originalIndex] = make_float4(worldPos.x, worldPos.y, worldPos.z, numberOfCollisionsTotal);
}

void markCollidingPoints(
        float* posOriginal,
        float* posSorted,
        unsigned int*  gridPointIndex,
        unsigned int*  pointCellStart,
        unsigned int*  pointCellStopp,
        unsigned int   numPoints)
{
    if(numPoints == 0) return;

    // thread per particle
    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numPoints, 128, numBlocks, numThreads);

    //std::cout << "markCollidingPoints(): we have " << numPoints << " points, " << numThreads << " threads and " << numBlocks << " blocks" << std::endl;

    // execute the kernel
    // TODO: test optimization: Write back not into posOriginal, but into posSorted (should be faster), then memcpy posSorted into posOriginal.
    // If writing back into posSorted is faster, we should be able to just switch buffers after every reduction.
    markCollidingPointsD<<< numBlocks, numThreads >>>(
                                               (float4*)posOriginal,
                                               (float4*)posSorted,
                                               gridPointIndex,
                                               pointCellStart,
                                               pointCellStopp,
                                               numPoints
                                               );

    hipDeviceSynchronize();

    cudaCheckSuccess("markCollidingPoints");
}

// bounding box type
typedef thrust::pair<float4, float4> bbox;

// reduce a pair of bounding boxes (a,b) to a bounding box containing a and b
struct bbox_reduction : public thrust::binary_function<bbox,bbox,bbox>
{
    __host__ __device__
    bbox operator()(bbox a, bbox b)
    {
        // min corner
        float4 min = make_float4(thrust::min(a.first.x, b.first.x), thrust::min(a.first.y, b.first.y), thrust::min(a.first.z, b.first.z), 0);

        // max corner
        float4 max = make_float4(thrust::max(a.second.x, b.second.x), thrust::max(a.second.y, b.second.y), thrust::max(a.second.z, b.second.z), 0);

        return bbox(min, max);
    }
};

// convert a point to a bbox containing that point, (point) -> (point, point)
struct bbox_transformation : public thrust::unary_function<float4,bbox>
{
    __host__ __device__
    bbox operator()(float4 point)
    {
        return bbox(point, point);
    }
};

void getBoundingBox(float *dPoints, uint numPoints, float3& min, float3& max)
{
    float4* points = (float4*)dPoints;

    // wrap raw pointer with a device_ptr
    thrust::device_ptr<float4> dev_ptr = thrust::device_pointer_cast(points);

    bbox init = bbox(dev_ptr[0], dev_ptr[0]);

    // transformation operation
    bbox_transformation opConvertPointToBoundingBox;

    // binary reduction operation
    bbox_reduction opUnifyBoundingBoxes;

    // compute the bounding box for the point set
    bbox result = thrust::transform_reduce(
                thrust::device_ptr<float4>(points),
                thrust::device_ptr<float4>(points + numPoints),
                opConvertPointToBoundingBox,
                init,
                opUnifyBoundingBoxes);

    min = make_float3(result.first);
    max = make_float3(result.second);
}

void sortMapAccordingToKeys(uint *dGridCellIndex, uint *dGridPointIndex, uint numPoints)
{
    if(numPoints == 0) return;

    thrust::sort_by_key(thrust::device_ptr<uint>(dGridCellIndex),                // KeysBeginning
                        thrust::device_ptr<uint>(dGridCellIndex + numPoints),    // KeysEnd
                        thrust::device_ptr<uint>(dGridPointIndex));              // ValuesBeginning

    cudaCheckSuccess("sortMapAccordingToKeys");
}

inline __host__ __device__ bool operator==(float4 a, float4 b)
{
    return
            a.x == b.x &&
            a.y == b.y &&
            a.z == b.z &&
            a.w == b.w;
}

unsigned int removeClearedPoints(float *devicePoints, unsigned int numberOfPoints)
{
    float4* points = (float4*)devicePoints;

    thrust::device_ptr<float4> newEnd;

    try
    {
        // Just for debugging!
        int result = thrust::count(thrust::device_ptr<float4>(points),
                                   thrust::device_ptr<float4>(points + numberOfPoints),
                                   make_float4(0.0f));
        std::cerr << __PRETTY_FUNCTION__ << " removing zero points: " << result << " of " << numberOfPoints << std::endl;

        newEnd = thrust::remove(
                    thrust::device_ptr<float4>(points),
                    thrust::device_ptr<float4>(points + numberOfPoints),
                    make_float4(0.0f)
                    );
    }
    catch(thrust::system_error &e)
    {
      // output an error message and exit
      std::cerr << "Error accessing vector element: " << e.what() << std::endl;
      exit(-1);
    }

    cudaCheckSuccess("removeZeroPoints");

    unsigned int numberOfPointsLeft = newEnd.get() - points;

    return numberOfPointsLeft;
}

struct IsOutsideBoundingBoxOp
{
    const float3 mBBoxMin, mBBoxMax;

    IsOutsideBoundingBoxOp(const float3& boxMin, const float3& boxMax) :
        mBBoxMin(boxMin),
        mBBoxMax(boxMax)
    { }

    __host__ __device__
    bool operator()(const float4 point)
    {
        return
                mBBoxMin.x > point.x ||
                mBBoxMin.y > point.y ||
                mBBoxMin.z > point.z ||
                mBBoxMax.x < point.x ||
                mBBoxMax.y < point.y ||
                mBBoxMax.z < point.z;
    }
};

unsigned int removePointsOutsideBoundingBox(float* points, unsigned int numberOfPoints, Grid* grid)
{
    printf("removePointsOutsideBoundingBox(): clearing %d points outside %.2f %.2f %.2f and %.2f %.2f %.2f\n", numberOfPoints, grid->worldMin.x, grid->worldMin.y, grid->worldMin.z, grid->worldMax.x, grid->worldMax.y, grid->worldMax.z);
    // move all points in bbox to beginning of devicePointsBase and return number of points left
    IsOutsideBoundingBoxOp op(grid->worldMin, grid->worldMax);

    float4* pointsf4 = (float4*)points;

    const thrust::device_ptr<float4> newEnd = thrust::remove_if(
                thrust::device_ptr<float4>(pointsf4),
                thrust::device_ptr<float4>(pointsf4 + numberOfPoints),
                op);

    cudaCheckSuccess("removePointsOutsideBoundingBox");

    unsigned int numberOfPointsRemaining = newEnd.get() - pointsf4;

    printf("removePointsOutsideBoundingBox(): done.\n");

    return numberOfPointsRemaining;
}

unsigned int copyPoints(float* devicePointsBaseDst, float* devicePointsBaseSrc, unsigned int numberOfPointsToCopy)
{
    float4* pointsSrc = (float4*)devicePointsBaseSrc;
    float4* pointsDst = (float4*)devicePointsBaseDst;

    const thrust::device_ptr<float4> newEnd = thrust::copy(
                thrust::device_ptr<float4>(pointsSrc),
                thrust::device_ptr<float4>(pointsSrc + numberOfPointsToCopy),
                thrust::device_ptr<float4>(pointsDst));

    cudaCheckSuccess("copyPoints");

    const unsigned int numberOfPointsCopied = newEnd.get() - pointsDst;
    return numberOfPointsCopied;
}


struct IsInsideBoundingBoxOp
{
    const float3 mBBoxMin, mBBoxMax;

    IsInsideBoundingBoxOp(const float3& boxMin, const float3& boxMax) :
        mBBoxMin(boxMin),
        mBBoxMax(boxMax)
    { }

    __host__ __device__
    bool operator()(const float4 point)
    {
        return
                mBBoxMin.x < point.x &&
                mBBoxMin.y < point.y &&
                mBBoxMin.z < point.z &&
                mBBoxMax.x > point.x &&
                mBBoxMax.y > point.y &&
                mBBoxMax.z > point.z;
    }
};

// Requires dst and src to live in device memory space
unsigned int copyPointsInBoundingBox(float* devicePointsBaseDst, float* devicePointsBaseSrc, float3 &bBoxMin, float3 &bBoxMax, unsigned int numberOfPointsToCopy)
{
    float4* pointsSrc = (float4*)devicePointsBaseSrc;
    float4* pointsDst = (float4*)devicePointsBaseDst;

    IsInsideBoundingBoxOp op(bBoxMin, bBoxMax);

    const thrust::device_ptr<float4> newEnd = thrust::copy_if(
                thrust::device_ptr<float4>(pointsSrc),
                thrust::device_ptr<float4>(pointsSrc + numberOfPointsToCopy),
                thrust::device_ptr<float4>(pointsDst),
                op);

    cudaCheckSuccess("copyPointsInBoundingBox");

    const unsigned int numberOfPointsCopied = newEnd.get() - pointsDst;
    return numberOfPointsCopied;
}
