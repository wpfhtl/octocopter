#include "hip/hip_runtime.h"
// Fix for gcc 4.7
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"
#include "thrust/remove.h"
#include "thrust/reduce.h"
#include "thrust/tuple.h"

#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include "pointcloud.cuh"

// 0 seems to be 10% faster with 256k particles
#define USE_TEX 0

#if USE_TEX
#define FETCH(t, i) tex1Dfetch(t##Tex, i)
#else
#define FETCH(t, i) t[i]
#endif


#if USE_TEX
// textures for particle position and velocity
texture<float4, 1, hipReadModeElementType> oldPointPosTex;
//texture<float4, 1, hipReadModeElementType> oldVelTex;

//texture<uint, 1, hipReadModeElementType> gridParticleHashTex;
texture<uint, 1, hipReadModeElementType> pointCellStartTex;
texture<uint, 1, hipReadModeElementType> pointCellStoppTex;
#endif

// pointcloud parameters in constant memory
__constant__ PointCloudParameters params;

// Calculate's a particle's containing cell in the uniform grid
__device__ int3 pcdCalcGridPos(float3 p)
{
    float3 cellSize;
    cellSize.x = (params.bBoxMax.x - params.bBoxMin.x) / params.gridSize.x;
    cellSize.y = (params.bBoxMax.y - params.bBoxMin.y) / params.gridSize.y;
    cellSize.z = (params.bBoxMax.z - params.bBoxMin.z) / params.gridSize.z;

    int3 gridPos;
    gridPos.x = floor((p.x - params.bBoxMin.x) / cellSize.x);
    gridPos.y = floor((p.y - params.bBoxMin.y) / cellSize.y);
    gridPos.z = floor((p.z - params.bBoxMin.z) / cellSize.z);
    return gridPos;
}

// Calculate a particle's hash value (=address in grid) from its containing cell (clamping to edges)
__device__ uint pcdCalcGridHash(int3 gridPos)
{
    gridPos.x = gridPos.x & (params.gridSize.x-1);  // wrap grid, assumes size is power of 2
    gridPos.y = gridPos.y & (params.gridSize.y-1);
    gridPos.z = gridPos.z & (params.gridSize.z-1);
    return __umul24(__umul24(gridPos.z, params.gridSize.y), params.gridSize.x) + __umul24(gridPos.y, params.gridSize.x) + gridPos.x;
}

// Calculate grid hash value for each particle
__global__
void computeMappingFromGridCellToPointD(
        uint*   gridCellIndex,  // output
        uint*   gridPointIndex, // output
        float4* pos,            // input: particle positions
        uint    numPoints)
{
    uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if(index >= numPoints) return;

    volatile float4 p = pos[index];

    // In which grid cell does the particle live?
    int3 gridPos = pcdCalcGridPos(make_float3(p.x, p.y, p.z));

    // Calculate the particle's hash from the grid-cell. This means particles in the same cell have the same hash
    uint hash = pcdCalcGridHash(gridPos);

    // This array is the key-part of the map, mapping cellId (=hash) to particleIndex. The term "map" is not
    // exactly correct, because there can be multiple keys (because one cell can store many particles)
    gridCellIndex[index] = hash;

    // It seems stupid to fill an array like "array[x]=x". But this array is the value-part of a map and will get sorted according to the keys (=gridParticleHash)
    gridPointIndex[index] = index;
}

// rearrange particle data into sorted order (sorted according to containing grid cell), and find the start of each cell in the sorted hash array
__global__
void sortPosAccordingToGridCellAndFillCellStartAndEndArraysD(
        uint*   pointCellStart,          // output: cell start index
        uint*   pointCellStopp,            // output: cell end index
        float4* sortedPos,          // output: sorted positions, sorted according to the containing gridcell
        uint *  gridCellIndex,      // input: sorted grid hashes
        uint *  gridParticleIndex,  // input: sorted particle indices
        float4* oldPointPos,             // input: UNsorted position array
        uint    numParticles)
{
    uint threadIndex = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;

    // This resides in shared memory space of the threadBlock, lives as
    // long as the block and is accessible from all threads in the block.
    // Its size (in bytes) is defined at runtime through the Ns parameter
    // in the <<Dg, Db, Ns, S>> expression of the caller.
    // Here, its set to ((ThreadsInBlock + 1) elements)
    extern __shared__ uint sharedHash[];

    uint hash;

    // When particleCount is smaller than a multiple of the block size, the remaining threads do nothing.
    if(threadIndex < numParticles)
    {
        hash = gridCellIndex[threadIndex];

        // Load hash data into shared memory so that we can look at neighboring
        // particle's hash value without loading two hash values per thread
        sharedHash[threadIdx.x+1] = hash; // => key of the sorted map

        if(threadIndex > 0 && threadIdx.x == 0)
        {
            // first thread in block must load neighbor particle hash
            sharedHash[0] = gridCellIndex[threadIndex-1];
        }
    }

    __syncthreads();

    if (threadIndex < numParticles)
    {
        // If this particle has a different cell index to the previous particle then it must be the
        // first particle in the cell, so store the index of this particle in the cell. As it isn't
        // the first particle, it must also be the cell end of the previous particle's cell
        if(threadIndex == 0 || hash != sharedHash[threadIdx.x])
        {
            pointCellStart[hash] = threadIndex;
            if (threadIndex > 0)
                pointCellStopp[sharedHash[threadIdx.x]] = threadIndex;
        }

        if(threadIndex == numParticles - 1)
        {
            pointCellStopp[hash] = threadIndex + 1;
        }

        // Now use the sorted index to reorder the pos and vel data
        uint sortedIndex = gridParticleIndex[threadIndex]; // => value of the sorted map
        float4 pos = FETCH(oldPointPos, sortedIndex);       // macro does either global read or texture fetch,
//        float4 vel = FETCH(oldVel, sortedIndex);       // see particles_kernel.cuh

        // ben: hier if() beenden, dann syncthreads() und dann nicht in sortedPos schreiben, sondern in oldPointPos? Bräuchte ich dann noch zwei pos/vel container?
        sortedPos[threadIndex] = pos;
//        sortedVel[threadIndex] = vel;
    }
}

// collide a particle against all other particles in a given cell
__device__
bool checkCellForNeighborsD(
        int3    gridPos,     // grid cell to search for particles that could collide
        uint    index,       // index of particle that is being collided
        float3  pos,         // position of particle that is being collided
        float4* oldPointPos,
        uint*   pointCellStart,
        uint*   pointCellStopp)
{
    uint gridHash = pcdCalcGridHash(gridPos);

    // get start of bucket for this cell
    uint startIndex = FETCH(pointCellStart, gridHash);

    // cell is not empty
    if(startIndex != 0xffffffff)
    {
        // iterate over particles in this cell
        uint endIndex = FETCH(pointCellStopp, gridHash);
        for(uint j=startIndex; j<endIndex; j++)
        {
            // check not colliding with self
            if (j != index)
            {
                float3 pos2 = make_float3(FETCH(oldPointPos, j));

                float3 relPos = pos - pos2;

                float dist = length(relPos);

                if(dist < params.minimumDistance)
                    return true;
            }
        }
    }
    return false;
}

// Collide a single particle (given by thread-id through @index) against all spheres in own and neighboring cells
__global__
void markCollidingPointsD(
        float4* posOriginal,     // output: new positions, same or zeroed. This is actually mDevicePointPos, so its the original position location
        float4* oldPointPos,          // input: positions sorted according to containing grid cell
        uint*   gridPointIndex,  // input: particle indices sorted according to containing grid cell
        uint*   pointCellStart,       // input: pointCellStart[19] contains the index of gridParticleIndex in which cell 19 starts
        uint*   pointCellStopp,         // input: pointCellStopp[19] contains the index of gridParticleIndex in which cell 19 ends
        uint    numPoints)       // input: number of total particles
{
    uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if(index >= numPoints) return;

    // read particle data from sorted arrays
    float3 pos = make_float3(FETCH(oldPointPos, index));
//    float3 vel = make_float3(FETCH(oldVel, index));

    // get address of particle in grid
    int3 gridPos = pcdCalcGridPos(pos);

    uint originalIndex = gridPointIndex[index];
    // examine neighbouring cells
    for(int z=-1; z<=1; z++)
    {
        for(int y=-1; y<=1; y++)
        {
            for(int x=-1; x<=1; x++)
            {
                int3 neighbourPos = gridPos + make_int3(x, y, z);
                if(
                        checkCellForNeighborsD(neighbourPos, index, pos, oldPointPos, pointCellStart, pointCellStopp)
                        &&
                        originalIndex % 2 == 0)
                {
                    // There is a neighboring point AND this point's index is even. Mark it for removal by zeroing it out!
                    posOriginal[originalIndex] = make_float4(0.0, 0.0, 0.0, 0.0);
                    return;
                }
            }
        }
    }

    // This point does not collide with any other. Do not change its values, it will be kept.
}


void setPointCloudParameters(PointCloudParameters *hostParams)
{
    // copy parameters to constant memory
    checkCudaSuccess("setPointCloudParameters(): CUDA error before const mem copy");
    hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(PointCloudParameters));
    checkCudaSuccess("setPointCloudParameters(): CUDA error after const mem copy");
}

// Calculates a hash for each particle. The hash value is ("based on") its cell id.
void computeMappingFromGridCellToPoint(
        uint*   gridCellIndex,
        uint*   gridPointIndex,
        float* pos,
        int     numPoints)
{
    uint numThreads, numBlocks;
    computeGridSize(numPoints, 256, numBlocks, numThreads);

    checkCudaSuccess("Kernel execution failed BEFORE computeMappingFromGridCellToPoint");

    // execute the kernel
    computeMappingFromGridCellToPointD<<< numBlocks, numThreads >>>(
                                                                      gridCellIndex,
                                                                      gridPointIndex,
                                                                      (float4*) pos,
                                                                      numPoints);

    // check if kernel invocation generated an error
    checkCudaSuccess("Kernel execution failed: computeMappingFromGridCellToPoint");
}

void sortPosAccordingToGridCellAndFillCellStartAndEndArrays(
        uint*  pointCellStart,
        uint*  pointCellStopp,
        float* sortedPos,
        uint*  gridCellIndex,
        uint*  gridPointIndex,
        float* oldPointPos,
        uint   numPoints,
        uint   numCells)
{
    checkCudaSuccess("sortPosAccordingToGridCellAndFillCellStartAndEndArrays(): cuda error present!");

    uint numThreads, numBlocks;
    computeGridSize(numPoints, 256, numBlocks, numThreads);

    // set all cells to empty
    hipMemset(pointCellStart, 0xffffffff, numCells*sizeof(uint));

    checkCudaSuccess("sortPosAccordingToGridCellAndFillCellStartAndEndArrays(): failed to set memory");

#if USE_TEX
    hipBindTexture(0, oldPointPosTex, oldPointPos, numPoints*sizeof(float4));
//    hipBindTexture(0, oldVelTex, oldVel, numPoints*sizeof(float4));
#endif

    checkCudaSuccess("sortPosAccordingToGridCellAndFillCellStartAndEndArrays(): failed to bind texture");

    // Number of bytes in shared memory that is allocated for each (thread)block.
    uint smemSize = sizeof(uint)*(numThreads+1);

    sortPosAccordingToGridCellAndFillCellStartAndEndArraysD<<< numBlocks, numThreads, smemSize>>>(
                                                                                                       pointCellStart,
                                                                                                       pointCellStopp,
                                                                                                       (float4*) sortedPos,
                                                                                                       gridCellIndex,
                                                                                                       gridPointIndex,
                                                                                                       (float4*) oldPointPos,
                                                                                                       numPoints);

    checkCudaSuccess("sortPosAccordingToGridCellAndFillCellStartAndEndArrays(): kernel failed");

#if USE_TEX
    hipUnbindTexture(oldPointPosTex);
//    hipUnbindTexture(oldVelTex);
#endif
}



void markCollidingPoints(
        float* posOriginal,
        float* posSorted,
        unsigned int*  gridPointIndex,
        unsigned int*  pointCellStart,
        unsigned int*  pointCellStopp,
        unsigned int   numPoints,
        unsigned int   numCells)
{
#if USE_TEX
    hipBindTexture(0, oldPointPosTex, posSorted, numPoints*sizeof(float4));
//    hipBindTexture(0, oldVelTex, sortedVel, numPoints*sizeof(float4));
    hipBindTexture(0, pointCellStartTex, pointCellStart, numCells*sizeof(uint));
    hipBindTexture(0, pointCellStoppTex, pointCellStopp, numCells*sizeof(uint));
#endif

    // thread per particle
    uint numThreads, numBlocks;
    computeGridSize(numPoints, 64, numBlocks, numThreads);

    // execute the kernel
    markCollidingPointsD<<< numBlocks, numThreads >>>(
                                               (float4*)posOriginal,
                                               (float4*)posSorted,
                                               gridPointIndex,
                                               pointCellStart,
                                               pointCellStopp,
                                               numPoints
                                               );

    // check if kernel invocation generated an error
    checkCudaSuccess("Kernel execution failed: markCollidingPoints");

#if USE_TEX
    hipUnbindTexture(oldPointPosTex);
//    hipUnbindTexture(oldVelTex);
    hipUnbindTexture(pointCellStartTex);
    hipUnbindTexture(pointCellStoppTex);
#endif
}



// bounding box type
typedef thrust::pair<float4, float4> bbox;

// reduce a pair of bounding boxes (a,b) to a bounding box containing a and b
struct bbox_reduction : public thrust::binary_function<bbox,bbox,bbox>
{
    __host__ __device__
    bbox operator()(bbox a, bbox b)
    {
        // min corner
        float4 min = make_float4(thrust::min(a.first.x, b.first.x), thrust::min(a.first.y, b.first.y), thrust::min(a.first.z, b.first.z), 0);

        // max corner
        float4 max = make_float4(thrust::max(a.second.x, b.second.x), thrust::max(a.second.y, b.second.y), thrust::max(a.second.z, b.second.z), 0);

        return bbox(min, max);
    }
};

// convert a point to a bbox containing that point, (point) -> (point, point)
struct bbox_transformation : public thrust::unary_function<float4,bbox>
{
    __host__ __device__
    bbox operator()(float4 point)
    {
        return bbox(point, point);
    }
};

void getBoundingBox(float *dPoints, uint numPoints, float3& min, float3& max)
{
    float4* points = (float4*)dPoints;

    // wrap raw pointer with a device_ptr
    thrust::device_ptr<float4> dev_ptr = thrust::device_pointer_cast(points);

    bbox init = bbox(dev_ptr[0], dev_ptr[0]);

    // initial bounding box contains first point - does this execute on host? If yes, how can dPoints[0] work?
//    bbox init = bbox(points[0], points[0]);
//    bbox init = bbox(thrust::device_ptr<float4>(dPoints)[0], thrust::device_ptr<float4>(dPoints)[0]);

    // transformation operation
    bbox_transformation opConvertPointToBoundingBox;

    // binary reduction operation
    bbox_reduction opUnifyBoundingBoxes;

    // compute the bounding box for the point set
    bbox result = thrust::transform_reduce(
                thrust::device_ptr<float4>(points),
                thrust::device_ptr<float4>(points + numPoints),
                opConvertPointToBoundingBox,
                init,
                opUnifyBoundingBoxes);

    min = make_float3(result.first);
    max = make_float3(result.second);
}

void sortMapAccordingToKeys(uint *dGridCellIndex, uint *dGridPointIndex, uint numPoints)
{
    checkCudaSuccess("Kernel execution failed BEFORE sortMapAccordingToKeys");

    thrust::sort_by_key(thrust::device_ptr<uint>(dGridCellIndex),                // KeysBeginning
                        thrust::device_ptr<uint>(dGridCellIndex + numPoints),    // KeysEnd
                        thrust::device_ptr<uint>(dGridPointIndex));              // ValuesBeginning

    checkCudaSuccess("Kernel execution failed: sortMapAccordingToKeys");
}

inline __host__ __device__ bool operator==(float4 a, float4 b)
{
    return
            a.x == b.x &&
            a.y == b.y &&
            a.z == b.z &&
            a.w == b.w;
}

unsigned int removeRedundantPoints(float *devicePoints, unsigned int numPoints)
{
    float4* points = (float4*)devicePoints;

    checkCudaSuccess("Kernel execution failed BEFORE removeRedundantPoints");

    const thrust::device_ptr<float4> newEnd = thrust::remove(thrust::device_ptr<float4>(points), thrust::device_ptr<float4>(points + numPoints), make_float4(0.0, 0.0, 0.0, 0.0));

    checkCudaSuccess("Kernel execution failed AFTER removeRedundantPoints");

    return newEnd.get() - points;
}
