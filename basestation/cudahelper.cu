#include "hip/hip_runtime.h"
#ifndef CUDA_CU
#define CUDA_CU

#include "cudahelper.cuh"

__device__ unsigned int getThreadIndex1D(void)
{
  return blockIdx.x * blockDim.x + threadIdx.x;
}


// compute the next higher power of 2 of 32-bit v
__host__ __device__ unsigned int nextHigherPowerOfTwo(unsigned int v)
{
    // decrements, then sets all bits below its most significant bit to 1, then it increments
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    return v + 1;
}

__host__ __device__ unsigned int cudaBound(const unsigned int min, const unsigned int value, const unsigned int max)
{
    if(value < min)
        return min;
    else if(value > max)
        return max;
    else
        return value;
}

#endif
