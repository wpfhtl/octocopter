#include "hip/hip_runtime.h"
// Fix for gcc 4.7
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <cuda_gl_interop.h>

#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particleskernel.cu"

void checkCudaSuccess(const char *errorMessage)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    printf("CUDA error %s: %s.\n", errorMessage, hipGetErrorString(err) );
    exit(-1);
  }
}

void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
{
    void *ptr;
    hipGraphicsMapResources(1, cuda_vbo_resource, 0);
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes, *cuda_vbo_resource);
    return ptr;
}

void copyArrayFromDevice(void* host, const void* device, struct hipGraphicsResource **cuda_vbo_resource, int size)
{
    if (cuda_vbo_resource) device = mapGLBufferObject(cuda_vbo_resource);

    hipMemcpy(host, device, size, hipMemcpyDeviceToHost);

    if (cuda_vbo_resource)
    {
        //unmapGLBufferObject(*cuda_vbo_resource);
        hipGraphicsUnmapResources(1, cuda_vbo_resource, 0);
    }
}

void setParameters(SimParams *hostParams)
{
    // copy parameters to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams));
}

//Round a / b to nearest higher integer value
uint iDivUp(uint a, uint b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

void integrateSystem(float *pos, float *vel, float deltaTime, uint numParticles)
{
    thrust::device_ptr<float4> d_pos4((float4 *)pos);
    thrust::device_ptr<float4> d_vel4((float4 *)vel);

    thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4)),
                thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_vel4+numParticles)),
                integrate_functor(deltaTime));
}

// Calculates a hash for each particle. The hash value is ("based on") its cell id.
void computeMappingFromGridCellToParticle(uint*  gridParticleHash,
              uint*  gridParticleIndex,
              float* pos,
              int    numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    computeMappingFromGridCellToParticleD<<< numBlocks, numThreads >>>(gridParticleHash,
                                           gridParticleIndex,
                                           (float4 *) pos,
                                           numParticles);

    // check if kernel invocation generated an error
    checkCudaSuccess("Kernel execution failed");
}

void sortPosAndVelAccordingToGridCellAndFillCellStartAndEndArrays(uint*  cellStart,
                                 uint*  cellEnd,
                                 float* sortedPos,
                                 float* sortedVel,
                                 uint*  gridParticleHash,
                                 uint*  gridParticleIndex,
                                 float* oldPos,
                                 float* oldVel,
                                 uint   numParticles,
                                 uint   numCells)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // set all cells to empty
    hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint));

#if USE_TEX
    hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4));
    hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4));
#endif

    // Number of bytes in shared memory that is allocated for each (thread)block.
    uint smemSize = sizeof(uint)*(numThreads+1);

    sortPosAndVelAccordingToGridCellAndFillCellStartAndEndArraysD<<< numBlocks, numThreads, smemSize>>>(
                                                                         cellStart,
                                                                         cellEnd,
                                                                         (float4 *) sortedPos,
                                                                         (float4 *) sortedVel,
                                                                         gridParticleHash,
                                                                         gridParticleIndex,
                                                                         (float4 *) oldPos,
                                                                         (float4 *) oldVel,
                                                                         numParticles);

    checkCudaSuccess("Kernel execution failed: reorderDataAndFindCellStartD");

#if USE_TEX
    hipUnbindTexture(oldPosTex);
    hipUnbindTexture(oldVelTex);
#endif
}

void collide(float* newVel,
             float* sortedPos,
             float* sortedVel,
             uint*  gridParticleIndex,
             uint*  cellStart,
             uint*  cellEnd,
             uint   numParticles,
             uint   numCells)
{
#if USE_TEX
    hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4));
    hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4));
    hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint));
    hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint));
#endif

    // thread per particle
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 64, numBlocks, numThreads);

    // execute the kernel
    collideD<<< numBlocks, numThreads >>>((float4*)newVel,
                                          (float4*)sortedPos,
                                          (float4*)sortedVel,
                                          gridParticleIndex,
                                          cellStart,
                                          cellEnd,
                                          numParticles);

    // check if kernel invocation generated an error
    checkCudaSuccess("Kernel execution failed");

#if USE_TEX
    hipUnbindTexture(oldPosTex);
    hipUnbindTexture(oldVelTex);
    hipUnbindTexture(cellStartTex);
    hipUnbindTexture(cellEndTex);
#endif
}

void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
{
    thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),                // KeysBeginning
                        thrust::device_ptr<uint>(dGridParticleHash + numParticles), // KeysEnd
                        thrust::device_ptr<uint>(dGridParticleIndex));              // ValuesBeginning
}
