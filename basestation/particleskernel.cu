#include "hip/hip_runtime.h"
// Fix for gcc 4.7
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#ifndef PARTICLES_KERNEL_H_
#define PARTICLES_KERNEL_H_

#include <stdio.h>
#include <math.h>
#include "cutil_math.h" // shouldn't be used, unsupported
#include "particleskernel.cuh"

#include "thrust/tuple.h"

#if USE_TEX
// textures for particle position and velocity
texture<float4, 1, hipReadModeElementType> oldPosTex;
texture<float4, 1, hipReadModeElementType> oldVelTex;

texture<uint, 1, hipReadModeElementType> gridParticleHashTex;
texture<uint, 1, hipReadModeElementType> cellStartTex;
texture<uint, 1, hipReadModeElementType> cellEndTex;
#endif

// simulation parameters in constant memory
__constant__ SimParams params;

struct integrate_functor
{
    float deltaTime;

    __host__ __device__
    integrate_functor(float delta_time) : deltaTime(delta_time) {}

    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        volatile float4 posData = thrust::get<0>(t);
        volatile float4 velData = thrust::get<1>(t);
        float3 pos = make_float3(posData.x, posData.y, posData.z);
        float3 vel = make_float3(velData.x, velData.y, velData.z);

        vel += params.gravity * deltaTime;
        vel *= params.globalDamping;

        // new position = old position + velocity * deltaTime
        pos += vel * deltaTime;

        // collisions with cube sides
        if (pos.x > params.worldMax.x - params.particleRadius) { pos.x = params.worldMax.x - params.particleRadius; vel.x *= params.boundaryDamping; }
        if (pos.x < params.worldMin.x + params.particleRadius) { pos.x = params.worldMin.x + params.particleRadius; vel.x *= params.boundaryDamping;}
        if (pos.y > params.worldMax.y - params.particleRadius) { pos.y = params.worldMax.y - params.particleRadius; vel.y *= params.boundaryDamping; }
        if (pos.y < params.worldMin.y + params.particleRadius) { pos.y = params.worldMin.y + params.particleRadius; vel.y *= params.boundaryDamping;}
        if (pos.z > params.worldMax.z - params.particleRadius) { pos.z = params.worldMax.z - params.particleRadius; vel.z *= params.boundaryDamping; }
        if (pos.z < params.worldMin.z + params.particleRadius) { pos.z = params.worldMin.z + params.particleRadius; vel.z *= params.boundaryDamping;}

        // store new position and velocity
        thrust::get<0>(t) = make_float4(pos, posData.w);
        thrust::get<1>(t) = make_float4(vel, velData.w);
    }
};

// calculate position in uniform grid
__device__ int3 calcGridPos(float3 p)
{
    int3 gridPos;
    gridPos.x = floor((p.x - params.worldMin.x) / params.cellSize.x);
    gridPos.y = floor((p.y - params.worldMin.y) / params.cellSize.y);
    gridPos.z = floor((p.z - params.worldMin.z) / params.cellSize.z);
    return gridPos;
}

// calculate address in grid from position (clamping to edges)
__device__ uint calcGridHash(int3 gridPos)
{
    gridPos.x = gridPos.x & (params.gridSize.x-1);  // wrap grid, assumes size is power of 2
    gridPos.y = gridPos.y & (params.gridSize.y-1);
    gridPos.z = gridPos.z & (params.gridSize.z-1);
    return __umul24(__umul24(gridPos.z, params.gridSize.y), params.gridSize.x) + __umul24(gridPos.y, params.gridSize.x) + gridPos.x;
}

// calculate grid hash value for each particle
__global__
void calcHashD(uint*   gridParticleHash,  // output
               uint*   gridParticleIndex, // output
               float4* pos,               // input: positions
               uint    numParticles)
{
    uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (index >= numParticles) return;

    volatile float4 p = pos[index];

    // get address in grid
    int3 gridPos = calcGridPos(make_float3(p.x, p.y, p.z));
    uint hash = calcGridHash(gridPos);

    // store grid hash and particle index
    gridParticleHash[index] = hash;
    gridParticleIndex[index] = index;
}

// rearrange particle data into sorted order, and find the start of each cell
// in the sorted hash array
__global__
void reorderDataAndFindCellStartD(uint*   cellStart,        // output: cell start index
                                  uint*   cellEnd,          // output: cell end index
                                  float4* sortedPos,        // output: sorted positions
                                  float4* sortedVel,        // output: sorted velocities
                                  uint *  gridParticleHash, // input: sorted grid hashes
                                  uint *  gridParticleIndex,// input: sorted particle indices
                                  float4* oldPos,           // input: sorted position array
                                  float4* oldVel,           // input: sorted velocity array
                                  uint    numParticles)
{
    uint threadIndex = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;

    // This resides in shared memory space of the threadBlock, lives as
    // long as the block and is accessible from all threads in the block.
    // Its size (in bytes) is defined at runtime through the Ns parameter
    // in the <<Dg, Db, Ns, S>> expression of the caller.
    // Here, its set to ((ThreadsInBlock + 1) elements)
    extern __shared__ uint sharedHash[];

    uint hash;
    // handle case when no. of particles not multiple of block size
    if(threadIndex < numParticles)
    {
        hash = gridParticleHash[threadIndex];

        // Load hash data into shared memory so that we can look at neighboring
        // particle's hash value without loading two hash values per thread
        sharedHash[threadIdx.x+1] = hash;

        if(threadIndex > 0 && threadIdx.x == 0)
        {
            // first thread in block must load neighbor particle hash
            sharedHash[0] = gridParticleHash[threadIndex-1];
        }
    }

    __syncthreads();

    if (threadIndex < numParticles)
    {
        // If this particle has a different cell index to the previous particle then it must be the
        // first particle in the cell, so store the index of this particle in the cell. As it isn't
        // the first particle, it must also be the cell end of the previous particle's cell
        if(threadIndex == 0 || hash != sharedHash[threadIdx.x])
        {
            cellStart[hash] = threadIndex;
            if (threadIndex > 0)
                cellEnd[sharedHash[threadIdx.x]] = threadIndex;
        }

        if(threadIndex == numParticles - 1)
        {
            cellEnd[hash] = threadIndex + 1;
        }

        // Now use the sorted index to reorder the pos and vel data
        uint sortedIndex = gridParticleIndex[threadIndex];
        float4 pos = FETCH(oldPos, sortedIndex);       // macro does either global read or texture fetch
        float4 vel = FETCH(oldVel, sortedIndex);       // see particles_kernel.cuh

        sortedPos[threadIndex] = pos;
        sortedVel[threadIndex] = vel;
    }
}

// collide two spheres using DEM method
__device__
float3 collideSpheres(float3 posA, float3 posB,
                      float3 velA, float3 velB,
                      float radiusA, float radiusB,
                      float attraction)
{
    // calculate relative position
    float3 relPos = posB - posA;

    float dist = length(relPos);
    float collideDist = radiusA + radiusB;

    float3 force = make_float3(0.0f);
    if (dist < collideDist)
    {
        float3 norm = relPos / dist;

        // relative velocity
        float3 relVel = velB - velA;

        // relative tangential velocity
        float3 tanVel = relVel - (dot(relVel, norm) * norm);

        // spring force
        force = -params.spring*(collideDist - dist) * norm;
        // dashpot (damping) force
        force += params.damping*relVel;
        // tangential shear force
        force += params.shear*tanVel;
        // attraction
        force += attraction*relPos;
    }

    return force;
}



// collide a particle against all other particles in a given cell
__device__
float3 collideCell(int3    gridPos,
                   uint    index,
                   float3  pos,
                   float3  vel,
                   float4* oldPos,
                   float4* oldVel,
                   uint*   cellStart,
                   uint*   cellEnd)
{
    uint gridHash = calcGridHash(gridPos);

    // get start of bucket for this cell
    uint startIndex = FETCH(cellStart, gridHash);

    float3 force = make_float3(0.0f);

    // cell is not empty
    if(startIndex != 0xffffffff)
    {
        // iterate over particles in this cell
        uint endIndex = FETCH(cellEnd, gridHash);
        for(uint j=startIndex; j<endIndex; j++)
        {
            // check not colliding with self
            if (j != index)
            {
                float3 pos2 = make_float3(FETCH(oldPos, j));
                float3 vel2 = make_float3(FETCH(oldVel, j));

                // collide two spheres
                force += collideSpheres(pos, pos2, vel, vel2, params.particleRadius, params.particleRadius, params.attraction);
            }
        }
    }
    return force;
}


__global__
void collideD(float4* newVel,               // output: new velocity
              float4* oldPos,               // input: sorted positions
              float4* oldVel,               // input: sorted velocities
              uint*   gridParticleIndex,    // input: sorted particle indices
              uint*   cellStart,
              uint*   cellEnd,
              uint    numParticles)
{
    uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if (index >= numParticles) return;

    // read particle data from sorted arrays
    float3 pos = make_float3(FETCH(oldPos, index));
    float3 vel = make_float3(FETCH(oldVel, index));

    // get address in grid
    int3 gridPos = calcGridPos(pos);

    // examine neighbouring cells
    float3 force = make_float3(0.0f);
    for(int z=-1; z<=1; z++) {
        for(int y=-1; y<=1; y++) {
            for(int x=-1; x<=1; x++) {
                int3 neighbourPos = gridPos + make_int3(x, y, z);
                force += collideCell(neighbourPos, index, pos, vel, oldPos, oldVel, cellStart, cellEnd);
            }
        }
    }

    // collide with cursor sphere
    force += collideSpheres(
                pos,
                params.colliderPos,
                vel,
                make_float3(0.0f, 0.0f, 0.0f),
                params.particleRadius,
                params.colliderRadius,
                0.0f);

    // write new velocity back to original unsorted location
    uint originalIndex = gridParticleIndex[index];
    newVel[originalIndex] = make_float4(vel + force, 0.0f);
}

#endif
