#include "hip/hip_runtime.h"
// Fix for gcc 4.7
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#ifndef PARTICLES_KERNEL_H_
#define PARTICLES_KERNEL_H_

#include "hip/hip_vector_types.h"
#include "particleskernel.cuh"

#include "thrust/tuple.h"

// simulation parameters in constant memory
__constant__ SimulationParameters params;

__device__ unsigned int getThreadIndex(void)
{
  return blockIdx.x * blockDim.x + threadIdx.x;
}


__device__ float3 getGridCellSize()
{
    return make_float3(
        (params.particleSystemWorldMax.x - params.particleSystemWorldMin.x) / params.particleSystemGridSize.x,
        (params.particleSystemWorldMax.y - params.particleSystemWorldMin.y) / params.particleSystemGridSize.y,
        (params.particleSystemWorldMax.z - params.particleSystemWorldMin.z) / params.particleSystemGridSize.z);
}

// Calculate's a particle's containing cell in the uniform grid
__device__ int3 getGridCellCoordinate(float3 worldPos)
{
    float3 cellSize = getGridCellSize();

    int3 gridPos;
    gridPos.x = floor((worldPos.x - params.particleSystemWorldMin.x) / cellSize.x);
    gridPos.y = floor((worldPos.y - params.particleSystemWorldMin.y) / cellSize.y);
    gridPos.z = floor((worldPos.z - params.particleSystemWorldMin.z) / cellSize.z);
    return gridPos;
}

// Calculate a particle's hash value (=address in grid) from its containing cell (clamping to edges)
__device__ uint getGridCellHash(int3 gridPos)
{
    gridPos.x = gridPos.x & (params.particleSystemGridSize.x-1);  // wrap grid, assumes size is power of 2
    gridPos.y = gridPos.y & (params.particleSystemGridSize.y-1);
    gridPos.z = gridPos.z & (params.particleSystemGridSize.z-1);
    return ((gridPos.z * params.particleSystemGridSize.y) * params.particleSystemGridSize.x) + (gridPos.y * params.particleSystemGridSize.x) + gridPos.x;
}

// Given the cell hash, whats the 3d-grid-coordinate of the cell's center?
// This is the reverse of calcGridHash(int3 gridCell).
__device__ uint3 getGridCellCoordinate(unsigned int hash)
{
    uint3 cell;
    cell.x = floor(fmod((double)hash, params.particleSystemGridSize.x));
    cell.y = floor(fmod((double)hash, params.particleSystemGridSize.x * params.particleSystemGridSize.y) / params.particleSystemGridSize.x);
    cell.z = floor(fmod((double)hash, params.particleSystemGridSize.x * params.particleSystemGridSize.y * params.particleSystemGridSize.z) / (params.particleSystemGridSize.x * params.particleSystemGridSize.y));
    return cell;
}


__device__ float3 getGridCellCenter(uint3 gridCellCoordinate)
{
    float3 cellSize = getGridCellSize();

    return make_float3(
                params.particleSystemWorldMin.x + (cellSize.x * gridCellCoordinate.x) + (cellSize.x / 2.0),
                params.particleSystemWorldMin.y + (cellSize.y * gridCellCoordinate.y) + (cellSize.y / 2.0),
                params.particleSystemWorldMin.z + (cellSize.z * gridCellCoordinate.z) + (cellSize.z / 2.0)
                );
}


// Used to copy the waypoint pressure from a uint8_t vector to the w-components of the cell-position-float4-vector.
// When extracting waypoints form device vectors, this is done, then the cell-positions are sorted DESC according
// to the w component, then the N first waypoints are extracted.
struct copy_functor
{
    __host__ __device__
    copy_functor() {}

    template <typename Tuple>
    //__host__ otherwise we get warnings that params (global mem) cannot be read directly in a host function
    __device__
    void operator()(Tuple t)
    {
        volatile uint8_t pressure = thrust::get<0>(t);
        volatile float4 cellWorldPosition = thrust::get<1>(t);

        // store new position and velocity
        thrust::get<1>(t) = make_float4(cellWorldPosition.x, cellWorldPosition.y, cellWorldPosition.z, pressure);
    }
};

// Integrate particles, same as above. But for setting the waypointPressureMap, we need out-of-order access that thrust::Tuple cannot provide
__global__
void integrateSystemD(
        float4*         particlePositions,          // in/out: particle positions
        float4*         particleVelocities,         // in/out: particle velocities
        unsigned char*  gridWaypointPressure,       // in/out: grid containing quint8-cells with waypoint-pressure values (80-255)
        float4*         particleCollisionPositions, // input:  particle positions
        float           deltaTime,
        uint            numParticles)
{
    const unsigned int index = getThreadIndex();
    if(index >= numParticles) return;

    float3 pos = make_float3(particlePositions[index].x, particlePositions[index].y, particlePositions[index].z);
    float3 vel = make_float3(particleVelocities[index].x, particleVelocities[index].y, particleVelocities[index].z);

    vel += params.gravity * deltaTime;
    vel *= params.dampingMotion;

    // If particle moves further than its radius in one iteration, it may slip through cracks that would be unpassable
    // in reality. To prevent this, do not move particles further than r in every timestemp
    float3 movement = vel * deltaTime;
    float safeParticleRadius = params.particleRadius * 0.9f;
    float distance = length(movement);
    if(distance >= safeParticleRadius)
    {
        vel /= distance / safeParticleRadius;
        movement = vel * deltaTime;
    }

    // new position = old position + velocity * deltaTime
    pos += movement;

    // collisions with cube sides
    if (pos.x > params.particleSystemWorldMax.x - params.particleRadius) { pos.x = params.particleSystemWorldMax.x - params.particleRadius; vel.x *= params.velocityFactorCollisionBoundary;}
    if (pos.x < params.particleSystemWorldMin.x + params.particleRadius) { pos.x = params.particleSystemWorldMin.x + params.particleRadius; vel.x *= params.velocityFactorCollisionBoundary;}
    if (pos.z > params.particleSystemWorldMax.z - params.particleRadius) { pos.z = params.particleSystemWorldMax.z - params.particleRadius; vel.z *= params.velocityFactorCollisionBoundary;}
    if (pos.z < params.particleSystemWorldMin.z + params.particleRadius) { pos.z = params.particleSystemWorldMin.z + params.particleRadius; vel.z *= params.velocityFactorCollisionBoundary;}
    if (pos.y > params.particleSystemWorldMax.y - params.particleRadius) { pos.y = params.particleSystemWorldMax.y - params.particleRadius; vel.y *= params.velocityFactorCollisionBoundary;}

    // special case: hitting bottom plane of bounding box
    if (pos.y < params.particleSystemWorldMin.y + params.particleRadius)
    {
        // put the particle back to the top, re-set velocity back to zero
        pos.y = params.particleSystemWorldMax.y - params.particleRadius;

        vel.x = (fmod((double)(index * vel.y) + pos.x, 65535.0) / 32768.0) - 1.0;
        vel.z = (fmod((double)(index * vel.x) + pos.z, 65535.0) / 32768.0) - 1.0;
        vel.y = 0.0f;


        // pcpData is the ParticleCollisionPosition, so a non-zero value means this particle has hit a collider and now reached the bottom.
        // Record this in gwpData and re-set the pcpData to zero.
        float3 lastCollisionPosition = make_float3(particleCollisionPositions[index].x, particleCollisionPositions[index].y, particleCollisionPositions[index].z);

        if(lastCollisionPosition.x != 0.0f || lastCollisionPosition.y != 0.0f || lastCollisionPosition.z != 0.0f)
        {
            // Find out in what cell the collision occured
            uint hash = getGridCellHash(getGridCellCoordinate(lastCollisionPosition));

            gridWaypointPressure[hash] = min(gridWaypointPressure[hash] + 1, 255);

            // Clear the particle's last position of collision
            particleCollisionPositions[index] = make_float4(0.0f);
        }
    }

    // store new position and velocity
    particlePositions[index] = make_float4(pos, /*posData.w*/1.0);
    particleVelocities[index] = make_float4(vel, /*velData.w*/1.0);
}

// Calculate grid hash value for each particle
__global__
void computeMappingFromGridCellToParticleD(
        uint*   gridParticleHash,  // output
        uint*   gridParticleIndex, // output
        float4* pos,               // input: particle positions
        uint    numParticles)
{
    const unsigned int index = getThreadIndex();
    if(index >= numParticles) return;

    volatile float4 p = pos[index];

    // In which grid cell does the particle live?
    int3 gridPos = getGridCellCoordinate(make_float3(p.x, p.y, p.z));

    // Calculate the particle's hash from the grid-cell. This means particles in the same cell have the same hash
    uint hash = getGridCellHash(gridPos);

    // This array is the key-part of the map, mapping cellId (=hash) to particleIndex. The term "map" is not
    // exactly correct, because there can be multiple keys (because one cell can store many particles)
    gridParticleHash[index] = hash;

    // It seems stupid to fill an array like "array[x]=x". But this array is the value-part of a map and will get sorted according to the keys (=gridParticleHash)
    gridParticleIndex[index] = index;
}

// rearrange particle data into sorted order (sorted according to containing grid cell), and find the start of each cell in the sorted hash array
__global__
void sortPosAndVelAccordingToGridCellAndFillCellStartAndEndArraysD(
        uint*   cellStart,         // output: cell start index
        uint*   cellEnd,           // output: cell end index
        float4* posSorted,         // output: sorted positions, sorted according to the containing gridcell
        float4* velSorted,         // output: sorted velocities, sorted according to the containing gridcell
        uint*   gridParticleHash,  // input:  sorted grid hashes
        uint*   gridParticleIndex, // input:  sorted particle indices
        float4* posUnsorted,       // input:  unsorted position array
        float4* velUnsorted,       // input:  unsorted velocity array
        uint    numParticles       // input:  number of particles/colliders
        )
{
    const unsigned int threadIndex = getThreadIndex();

    // This resides in shared memory space of the threadBlock, lives as
    // long as the block and is accessible from all threads in the block.
    // Its size (in bytes) is defined at runtime through the Ns parameter
    // in the <<Dg, Db, Ns, S>> expression of the caller.
    // Here, its set to ((ThreadsInBlock + 1) elements)
    extern __shared__ uint sharedHash[];

    uint hash;

    // When particleCount is smaller than a multiple of the block size, the remaining threads do nothing.
    if(threadIndex < numParticles)
    {
        hash = gridParticleHash[threadIndex];

        // Load hash data into shared memory so that we can look at neighboring
        // particle's hash value without loading two hash values per thread
        sharedHash[threadIdx.x+1] = hash; // => key of the sorted map

        if(threadIndex > 0 && threadIdx.x == 0)
        {
            // first thread in block must load neighbor particle hash
            sharedHash[0] = gridParticleHash[threadIndex-1];
        }
    }

    __syncthreads();

    if (threadIndex < numParticles)
    {
        // If this particle has a different cell index to the previous particle then it must be the
        // first particle in the cell, so store the index of this particle in the cell. As it isn't
        // the first particle, it must also be the cell end of the previous particle's cell
        if(threadIndex == 0 || hash != sharedHash[threadIdx.x])
        {
            cellStart[hash] = threadIndex;
            if (threadIndex > 0)
                cellEnd[sharedHash[threadIdx.x]] = threadIndex;
        }

        if(threadIndex == numParticles - 1)
        {
            cellEnd[hash] = threadIndex + 1;
        }

        // Now use the sorted index to reorder the pos and vel data
        uint sortedIndex = gridParticleIndex[threadIndex]; // => value of the sorted map
        float4 pos, vel;

        // Only use vel if passed vels are non-zero. This way, we can use this method also for colliders, which have no velocities.
        pos = posUnsorted[sortedIndex];
        if(velUnsorted && velSorted)
            vel = velUnsorted[sortedIndex];

        // ben: hier if() beenden, dann syncthreads() und dann nicht in sortedPos schreiben, sondern in oldPos? Bräuchte ich dann noch zwei pos/vel container?
        posSorted[threadIndex] = pos;
        if(velUnsorted && velSorted) velSorted[threadIndex] = vel;
    }
}

// collide two spheres using DEM method
__device__
float3 collideSpheres(
        float3 posParticle,
        float3 velParticle,
        float radiusParticle,
        float3 posCollider,
        float3 velCollider,
        float radiusCollider,
        float attraction)
{
    // calculate relative position
    float3 relPos = posCollider - posParticle;

    float distance = length(relPos);
    float collisionDistance = radiusParticle + radiusCollider;

    float3 force = make_float3(0.0f);
    if (distance < collisionDistance)
    {
        float3 normal = relPos / distance;

        // relative velocity
        float3 relVel = velCollider - velParticle;

        // relative tangential velocity
        float3 tanVel = relVel - (dot(relVel, normal) * normal);

        // spring force
        force = params.spring * (collisionDistance - distance) * normal;

        // dashpot (damping) force
        force += params.velocityFactorCollisionParticle*relVel;

        // tangential shear force
        force += params.shear*tanVel;

        // attraction
        force += attraction*relPos;
    }

    return force;
}


// collide a particle against all other particles and colliders in a given cell
__device__
float3 collideCell(
        float4* particleCollisionPositions, // output: storage for the particle's current position if it collides with a collider
        int3    gridCellToSearch,       // input: grid cell to search for particles that could collide
        uint    particleToCollideIndex, // input: index of particle that is being collided
        float3  particleToCollidePos,   // input: position of particle that is being collided
        float3  particleToCollideVel,   // input: velocity of particle that is being collided

        float4* particlePosSorted,      // input: sorted positions  of particles to collide with
        float4* particleVelSorted,      // input: sorted velocities of particles to collide with
        uint*   particleCellStart,      // input: cellStart[x] gives us the index of particle[Pos|Vel]Sorted in which the particles in cell x start
        uint*   particleCellEnd,        // input: cellEnd  [x] gives us the index of particle[Pos|Vel]Sorted in which the particles in cell x end

        float4* colliderPosSorted,      // input: sorted positions of colliders to collide with
        uint*   colliderCellStart,      // input: cellStart[x] gives us the index of colliderPosSorted in which the colliders in cell x start
        uint*   colliderCellEnd         // input: cellEnd  [x] gives us the index of colliderPosSorted in which the colliders in cell x end
        )
{
    uint gridHash = getGridCellHash(gridCellToSearch);

    float3 forceCollisionsAgainstParticles = make_float3(0.0f);

    // Collide against other particles. Get start of bucket for this cell
    uint particlesStartIndex = particleCellStart[gridHash];
    // cell is not empty
    if(particlesStartIndex != 0xffffffff)
    {
        // iterate over particles in this cell
        uint particlesEndIndex = particleCellEnd[gridHash];

        for(uint j=particlesStartIndex; j<particlesEndIndex; j++)
        {
            // check not colliding with self
            if (j != particleToCollideIndex)
            {
                float3 posToCollideAgainst = make_float3(particlePosSorted[j]);
                float3 velToCollideAgainst = make_float3(particleVelSorted[j]);

                // collide two spheres
                forceCollisionsAgainstParticles += collideSpheres(
                            particleToCollidePos,
                            particleToCollideVel,
                            params.particleRadius,
                            posToCollideAgainst,
                            velToCollideAgainst,
                            params.particleRadius,
                            params.attraction);
            }
        }
    }

    float3 forceCollisionsAgainstColliders = make_float3(0.0f);

    // Collide against other particles. Get start of bucket for this cell
    uint collidersStartIndex = colliderCellStart[gridHash];
    // cell is not empty
    if(collidersStartIndex != 0xffffffff)
    {
        // iterate over particles in this cell
        uint collidersEndIndex = colliderCellEnd[gridHash];

        for(uint j=collidersStartIndex; j<collidersEndIndex; j++)
        {
            float3 posToCollideAgainst = make_float3(colliderPosSorted[j]);

            // collide two spheres
            forceCollisionsAgainstColliders += collideSpheres(
                        particleToCollidePos,
                        particleToCollideVel,
                        params.particleRadius,
                        posToCollideAgainst,
                        make_float3(0.0f),
                        0.1f, // radius of collider
                        params.attraction);
        }
    }

    // If the particle collided with a collider, store its current position
    if(forceCollisionsAgainstColliders.x != 0.0f || forceCollisionsAgainstColliders.y != 0.0f || forceCollisionsAgainstColliders.z != 0.0f)
    {
        // Store the particle's last collision-position. When the particle reaches the bottom-plane,
        // integrateSystem() increments the value of the cell that last collision appeared in.
        particleCollisionPositions[particleToCollideIndex] = make_float4(particleToCollidePos, 0.0);
        //return make_float3(0.0f, 0.0f, 0.0f);
    }

    return forceCollisionsAgainstParticles + forceCollisionsAgainstColliders;
}


// Collide a single particle (given by thread-id through @index) against all other particles and colliders in own and neighboring cells
__global__
void collideParticlesWithParticlesAndCollidersD(
        float4* newVel,             // output: new velocities. This is actually mDeviceVel, so its the original velocity location
        float4* particleCollisionPositions,          // output: Every particle's position of last collision, or 0.0/0.0/0.0 if none occurred.

        float4* particlePosSorted,  // input: particle positions sorted according to containing grid cell
        float4* particleVelSorted,  // input: particle velocities sorted according to containing grid cell
        uint*   particleMapIndex,   // input: particle indices sorted according to containing grid cell
        uint*   particleCellStart,  // input: cellStart[19] contains the index of gridParticleIndex in which particles in cell 19 start
        uint*   particleCellEnd,    // input: cellEnd[19] contains the index of gridParticleIndex in which particles in cell 19 end

        float4* colliderPosSorted,  // input: collider positions sorted according to containing grid cell
        uint*   colliderMapIndex,   // input: collider indices sorted according to containing grid cell
        uint*   colliderCellStart,  // input: cellStart[19] contains the index of gridColliderIndex in which colliders in cell 19 start
        uint*   colliderCellEnd,    // input: cellEnd[19] contains the index of gridColliderIndex in which colliders in cell 19 end

        uint    numParticles)       // input: number of total particles
{
    uint particleToCollideIndex = getThreadIndex();
    if (particleToCollideIndex >= numParticles) return;

    // read particle data from sorted arrays
    float3 particleToCollidePos = make_float3(particlePosSorted[particleToCollideIndex]);
    float3 particleToCollideVel = make_float3(particleVelSorted[particleToCollideIndex]);

    // get grid-cell of particle
    int3 particleToCollideGridCell = getGridCellCoordinate(particleToCollidePos);

    // examine neighbouring cells
    float3 forceOnParticle = make_float3(0.0f);

    for(int z=-1; z<=1; z++)
    {
        for(int y=-1; y<=1; y++)
        {
            for(int x=-1; x<=1; x++)
            {
                int3 neighbourGridCell = particleToCollideGridCell + make_int3(x, y, z);

                // Collide against other particles and colliders in this cell
                forceOnParticle += collideCell(
                            particleCollisionPositions,
                            neighbourGridCell,
                            particleToCollideIndex,
                            particleToCollidePos,
                            particleToCollideVel,

                            particlePosSorted,
                            particleVelSorted,
                            particleCellStart,
                            particleCellEnd,

                            colliderPosSorted,
                            colliderCellStart,
                            colliderCellEnd);
            }
        }
    }

    // write new velocity back to original unsorted location
    uint originalIndex = particleMapIndex[particleToCollideIndex];
    newVel[originalIndex] = make_float4(particleToCollideVel + forceOnParticle, 0.0f);
}

__global__
void fillGridMapCellWorldPositionsD(
        float4* gridMapCellWorldPositions,
        uint numberOfCells)
{
    uint cellIndex = getThreadIndex();
    if(cellIndex >= numberOfCells) return;

    float3 gridCellCoordinate = make_float3(
                floor(fmod((double)cellIndex, (double)(params.particleSystemGridSize.x))),
                floor(fmod((double)cellIndex, (double)(params.particleSystemGridSize.x * params.particleSystemGridSize.y)) / params.particleSystemGridSize.x),
                floor(fmod((double)cellIndex, (double)(params.particleSystemGridSize.x * params.particleSystemGridSize.y * params.particleSystemGridSize.z)) / (params.particleSystemGridSize.x * params.particleSystemGridSize.y))
                );

    float3 cellSize;
    cellSize.x = (params.particleSystemWorldMax.x - params.particleSystemWorldMin.x) / params.particleSystemGridSize.x;
    cellSize.y = (params.particleSystemWorldMax.y - params.particleSystemWorldMin.y) / params.particleSystemGridSize.y;
    cellSize.z = (params.particleSystemWorldMax.z - params.particleSystemWorldMin.z) / params.particleSystemGridSize.z;

    gridMapCellWorldPositions[cellIndex] = make_float4(
                params.particleSystemWorldMin.x + (cellSize.x * gridCellCoordinate.x) + (cellSize.x / 2.0),
                params.particleSystemWorldMin.y + (cellSize.y * gridCellCoordinate.y) + (cellSize.y / 2.0),
                params.particleSystemWorldMin.z + (cellSize.z * gridCellCoordinate.z) + (cellSize.z / 2.0),
                0.0f
                );
}

// For the life of me, I cannot figure out why this kernel gives an "unspecified launch failure".
// Reverting to serial host-only code for now :(
__global__
void moveGridMapWayPointPressureValuesByWorldPositionOffsetD(
        uint8_t* gridMapOfWayPointPressure,
        float3* offset,
        uint numberOfCells)
{
    uint cellIndex = getThreadIndex();
    if(cellIndex >= numberOfCells) return;

    uint8_t pressure = gridMapOfWayPointPressure[cellIndex];

    __syncthreads();

    uint3 gridCellCoordinate = getGridCellCoordinate(cellIndex);

    float3 gridCellWorldPosition = getGridCellCenter(gridCellCoordinate);

    uint newIndex = getGridCellHash(getGridCellCoordinate(gridCellWorldPosition + *offset));

    if(newIndex < numberOfCells) gridMapOfWayPointPressure[newIndex] = pressure;
}

#endif
