#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <QDebug>

// Kernel definition
extern "C" void computeFreeColumns(unsigned char* gridPointer, unsigned char* pixmap, int x, int y, int z);

__global__ void kernelComputeFreeColumns(unsigned char* gridPointer, unsigned char* pixmap, int x, int y, int z)
{
int i = threadIdx.x;
int j = threadIdx.y;

//pixmap[i] = i*j;//gridPointer[i];
//pixmap[i] = i*j;//gridPointer[i];

i = blockDim.x * blockIdx.x + threadIdx.x;

//gridPointer[i] = i;
pixmap[i] = gridPointer[i];
}

void computeFreeColumns(unsigned char* grid, unsigned char* pixmap, int x, int y, int z)
{
    hipError_t mCudaError;

    //    int numBlocks = 1;
    //    dim3 threadsPerBlock(N, N);
    //    MatAdd<<<numBlocks, threadsPerBlock>>>(A, B, C);

    // gridPointer points to the host's grid. We need the pointer to the device's version of that data.
    quint8* deviceVolumeDataBasePointer = 0;
    mCudaError = hipHostGetDevicePointer(&deviceVolumeDataBasePointer, grid, 0);
    if(deviceVolumeDataBasePointer == 0) qDebug("couldn't get device pointer for volume data");
    if(mCudaError != hipSuccess) qDebug("couldn't get device pointer for volume data: %s", hipGetErrorString(mCudaError));

    unsigned char *pixmapDevice;
    hipMalloc((void**)&pixmapDevice, x*z);

    kernelComputeFreeColumns<<<16, 64>>>(deviceVolumeDataBasePointer, pixmapDevice, x, y, z);

    hipMemcpy(pixmap, pixmapDevice, x*z, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipDeviceSynchronize();

    qDebug() << "grid:";
    for(int i=0;i<(x*y*z)/8;i++) printf("%d ", grid[i]);
    fflush(stdout);

    qDebug() << "\npixmap:";
    for(int i=0;i<x*z;i++) printf("%d ", pixmap[i]);
    fflush(stdout);

    hipFree(pixmapDevice);
}



