#include "hip/hip_runtime.h"
// Fix for gcc 4.7
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include "grid.cuh"
#include "cudahelper.cuh"
#include "hip/hip_vector_types.h"

#include "parameterspathplanner.cuh"

// simulation parameters in constant memory
__constant__ ParametersPathPlanner parametersPathPlanner;

void copyParametersToGpu(ParametersPathPlanner *hostParams)
{
    // Copy parameters to constant memory. This was synchronous once, I changed
    // it to be asynchronous. Shouldn't cause any harm, even if parameters were
    // applied one frame too late.
    cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(parametersPathPlanner), hostParams, sizeof(ParametersPathPlanner)));
}

__global__
void fillOccupancyGridD(unsigned char* gridValues, float4* colliderPos, unsigned int numColliders)
{
    uint particleIndex = getThreadIndex1D();
    if (particleIndex >= numColliders) return;

    float3 particleToCollidePos = make_float3(colliderPos[particleIndex]);

    // get grid-cell of particle
    int3 particleGridCell = parametersPathPlanner.grid.getCellCoordinate(particleToCollidePos);

    // The cell-hash IS the offset in memory, as cells are adressed linearly
    uint cellHash = parametersPathPlanner.grid.getCellHash(particleGridCell);

    gridValues[cellHash] = 255;
}

void fillOccupancyGrid(unsigned char* gridValues, float* colliderPos, unsigned int numColliders, unsigned int numCells, hipStream_t *stream)
{
    if(numColliders == 0) return;

    // set all cells to empty
    hipMemset(gridValues, 0, numCells * sizeof(unsigned char));

    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numColliders, 64, numBlocks, numThreads);

    // Number of bytes in shared memory that is allocated for each (thread)block.
//    uint smemSize = sizeof(uint)*(numThreads+1);

    fillOccupancyGridD<<< numBlocks, numThreads, 0, *stream>>>(gridValues, (float4*)colliderPos, numColliders);

    cudaCheckSuccess("fillOccupancyGrid");
}



__device__
void growCellsD(unsigned char* gridValues, unsigned int numCells)
{
    uint cellIndex = getThreadIndex1D();
    if (cellIndex >= numCells) return;

    // get grid-cell of particle
    int3 threadGridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(cellIndex);

    unsigned int lowestNonNullNeighbor = 1000; // higher than u_int8_t could ever be
    unsigned int ownValue = gridValues[cellIndex];

    // When growing, we do not use the 26 3d-neighbors, but only 6 (up/down, left/right, front/back).
    // This means that we do not grow diagonally, and later do not travel diagonally. Thats better as
    // it keep us from going diagonally between two occupied cells.
    /*for(int z=-1; z<=1; z++)
    {
        for(int y=-1; y<=1; y++)
        {
            for(int x=-1; x<=1; x++)
            {
                int3 neighbourGridCellCoordinate = threadGridCellCoordinate + make_int3(x, y, z);

                uint neighbourGridCellOffset = parametersPathPlanner.grid.getCellHash(neighbourGridCellCoordinate);

                if(gridValues[neighbourGridCellOffset] < lowestNonNullNeighbor && gridValues[neighbourGridCellOffset] != 0)
                    lowestNonNullNeighbor = gridValues[neighbourGridCellOffset];
            }
        }
    }*/

    int3 neighbors[6];
    neighbors[0] = make_int3(1, 0, 0);
    neighbors[1] = make_int3(-1, 0, 0);
    neighbors[2] = make_int3(0, 1, 0);
    neighbors[3] = make_int3(0, -1, 0);
    neighbors[4] = make_int3(0, 0, 1);
    neighbors[5] = make_int3(0, 0, -1);

    for(int i=0;i<6;i++)
    {
        int3 neighbourGridCellCoordinate = threadGridCellCoordinate + neighbors[i];

        uint neighbourGridCellOffset = parametersPathPlanner.grid.getCellHash(neighbourGridCellCoordinate);

        if(gridValues[neighbourGridCellOffset] < lowestNonNullNeighbor && gridValues[neighbourGridCellOffset] != 0)
            lowestNonNullNeighbor = gridValues[neighbourGridCellOffset];
    }

    // Overwrite our cell's value (0 or the lowest distance to "start" Write into our cell if
    if(lowestNonNullNeighbor != 1000 && lowestNonNullNeighbor != 255 && (lowestNonNullNeighbor + 1) < ownValue)
        gridValues[cellIndex] = lowestNonNullNeighbor + 1;
}

__global__
void computePathD(unsigned char* gridValues, unsigned int numCells)
{
    uint cellIndex = getThreadIndex1D();

    int3 goalGridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(parametersPathPlanner.goal);
    uint goalGridCellOffset = parametersPathPlanner.grid.getCellHash(goalGridCellCoordinate);

    // Only act if the goal is not occupied and don't use more threads than cells
    if(gridValues[goalGridCellOffset] == 0 && cellIndex < numCells)
    {
        // Let the first thread set the cell containing "start" to 1!
        if(cellIndex == 0)
        {
            int3 startGridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(parametersPathPlanner.start);
            uint startGridCellOffset = parametersPathPlanner.grid.getCellHash(startGridCellCoordinate);
            gridValues[startGridCellOffset] = 1;
        }

        uint numIterations = 0;
        do
        {
            growCellsD(gridValues, numCells);
            numIterations++;
        }
        while(gridValues[goalGridCellOffset] == 0 && numIterations < 512);
    }
}

__global__
void retrievePathD(unsigned char* gridValues, float4* waypoints)
{
    int3 goalGridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(parametersPathPlanner.goal);
    uint goalGridCellOffset = parametersPathPlanner.grid.getCellHash(goalGridCellCoordinate);

    uint valueInGoalCell = gridValues[goalGridCellOffset];

    if(valueInGoalCell == 0)
    {
        // Tell the caller we failed to find a valid path by setting the first waypoint to all-zero.
        waypoints[0] = make_float4(0.0, 0.0, 0.0, 0.0);
    }
    else if(valueInGoalCell == 255)
    {
        // Tell the caller we failed to find a valid path because of an occupied target cell.
        waypoints[0] = make_float4(0.0, 0.0, 0.0, 1.0);
    }
    else
    {
        // Use this ONE thread to collect all the waypoints. The first float4 will contain
        // the number of waypoints including start and goal. The next float4s will be those
        // waypoints. Add 0.1 so we can cast to int without losing something.
        waypoints[0] = make_float4(valueInGoalCell + 0.1);

        // Set the last waypoint, which equals the goal position
        waypoints[valueInGoalCell] = make_float4(parametersPathPlanner.goal);

        // Now traverse from goal back to start and save the world positions in waypoints
        uint stepsToStartCell = valueInGoalCell;
        int3 cellCoordinate = goalGridCellCoordinate;

        do
        {
            // We are at cellCoordinate and found a value of distance. Now check all neighbors
            // until we find one with a smaller value. Thats the path backwards towards the goal.

            bool foundNextCellTowardsTarget = false;

            // See above for reasons for using only a 6-cell-neighborhood
            int3 neighbors[6];
            neighbors[0] = make_int3(1, 0, 0);
            neighbors[1] = make_int3(-1, 0, 0);
            neighbors[2] = make_int3(0, 1, 0);
            neighbors[3] = make_int3(0, -1, 0);
            neighbors[4] = make_int3(0, 0, 1);
            neighbors[5] = make_int3(0, 0, -1);

            for(int i=0; i<6 && !foundNextCellTowardsTarget; i++)
            {
                int3 neighbourCellCoordinate = cellCoordinate + neighbors[i];

                uint neighbourCellOffset = parametersPathPlanner.grid.getCellHash(neighbourCellCoordinate);

                u_int8_t neighborValue = gridValues[neighbourCellOffset];

                if(neighborValue < stepsToStartCell)
                {
                    // We found a neighbor with a smaller distance. Use it!
                    cellCoordinate = neighbourCellCoordinate;

                    // Append our current cell's position to the waypoint list.
                    waypoints[neighborValue] = make_float4(parametersPathPlanner.grid.getCellCenter(cellCoordinate));

                    // Escape those 3 for-loops to continue searching from this next cell.
                    foundNextCellTowardsTarget = true;

                    // Update distance to start-position, should be a simple decrement.
                    stepsToStartCell = neighborValue;
                }
            }

            /*for(int z=-1; z<=1 && !foundNextCellTowardsTarget; z++)
            {
                for(int y=1; y>=-1 && !foundNextCellTowardsTarget; y--) // try the higher cells first!
                {
                    for(int x=-1; x<=1 && !foundNextCellTowardsTarget; x++)
                    {
                        int3 neighbourCellCoordinate = cellCoordinate + make_int3(x, y, z);

                        uint neighbourCellOffset = parametersPathPlanner.grid.getCellHash(neighbourCellCoordinate);

                        uint neighborValue = gridValues[neighbourCellOffset];

                        if(neighborValue < stepsToStartCell)
                        {
                            // We found a neighbor with a smaller distance. Use it!
                            cellCoordinate = neighbourCellCoordinate;

                            // Append our current cell's position to the waypoint list.
                            waypoints[neighborValue - 1] = make_float4(parametersPathPlanner.grid.getCellCenter(cellCoordinate));

                            // Escape those 3 for-loops to continue searching from this next cell.
                            foundNextCellTowardsTarget = true;

                            // Update distance to start-position, should be a simple decrement.
                            stepsToStartCell = neighborValue;
                        }
                    }
                }
            }*/
        }
        while(stepsToStartCell > 1);

        // waypoints[1] was filled above with the cell-center. But we want it to be the start-position, which
        // - although contained in the cell - is probably not exactly its center.
        waypoints[1] = make_float4(parametersPathPlanner.start);
    }
}

void computePath(unsigned char* gridValues, unsigned int numCells, float *waypoints, hipStream_t *stream)
{
    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numCells, 64, numBlocks, numThreads);
    computePathD<<< numBlocks, numThreads, 0, *stream>>>(gridValues, numCells);

    retrievePathD<<< 1, 1, 0, *stream>>>(gridValues, (float4*)waypoints);
}
