#include "hip/hip_runtime.h"
// Fix for gcc 4.7
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include "grid.cuh"
#include "cudahelper.cuh"
#include "hip/hip_vector_types.h"

#include "parameterspathplanner.cuh"
#include "pathplanner.cuh"

// only for printf debugging
#include <stdlib.h>
#include <stdio.h>

// simulation parameters in constant memory
__constant__ ParametersPathPlanner parametersPathPlanner;
__constant__ Grid growingGrid;

void copyParametersToGpu(ParametersPathPlanner *hostParams)
{
    // Copy parameters to constant memory.
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(parametersPathPlanner), hostParams, sizeof(ParametersPathPlanner)));
}

__global__
void fillOccupancyGridD(u_int8_t* gridValues, float4* colliderPos, unsigned int numColliders)
{
    uint particleIndex = getThreadIndex1D();

    if(particleIndex >= numColliders) return;

    // ignore points scanned from closer than 3 meters, so that we ignore bernd and the fishing rod :)
    float4 particleToCollidePos = colliderPos[particleIndex];

    //if(particleToCollidePos.w > 9.0) // 3m squared
    //{
        // get grid-cell of particle
        int3 particleGridCell = parametersPathPlanner.grid.getCellCoordinate(make_float3(particleToCollidePos));

        // The cell-hash IS the offset in memory, as cells are adressed linearly
        int cellHash = parametersPathPlanner.grid.getSafeCellHash(particleGridCell);

        if(cellHash >= 0) gridValues[cellHash] = 255;
    //}
}

__global__
void dilateOccupancyGridD(u_int8_t* gridValues, unsigned int numCells)
{
    // Dilate the occupied cells for additional safety. This also allows expanding routes diagonally
    // later-on, as its ok to pass diagonally between occupied cells after dilation
    uint cellIndex = getThreadIndex1D();
    if(cellIndex >= numCells) return;

    u_int8_t ownValue = gridValues[cellIndex];
    if(ownValue == 255) return;

    int3 threadGridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(cellIndex);

    for(int z=-1;z<=1;z++)
    {
        for(int y=-1;y<=1;y++)
        {
            for(int x=-1;x<=1;x++)
            {
                const int3 neighbourGridCellCoordinate = threadGridCellCoordinate + make_int3(x,y,z);
                //if(cellIndex == 0) printf("cellIndex 0, coord 0/0/0 neighbor %d/%d/%d\n", x, y, z);

                if(parametersPathPlanner.grid.isCellInGrid(neighbourGridCellCoordinate))
                {
                    const int neighbourGridCellIndex = parametersPathPlanner.grid.getSafeCellHash(neighbourGridCellCoordinate);
                    // Because CUDA works using thread-batches, we cannot just load all cells, then compute and then store all cells.
                    // Using batches would mean we would dilate a part of the grid, then load the neighboring part and dilate the
                    // dilation again, making almost all of the grid become occupied.
                    // For this reason, we say that 255 is occupied and 254 is dilated-occupied. This way, we don't need two grids. Hah!
                    if(gridValues[neighbourGridCellIndex] == 255)
                    {
                        gridValues[cellIndex] = 254;
                        return;
                    }
                }
            }
        }
    }
}

__global__ void clearOccupancyGridAboveVehiclePositionD(
        unsigned char* gridValues,
        float vehicleX,
        float vehicleY,
        float vehicleZ)
{
    float3 vehiclePos = make_float3(vehicleX, vehicleY, vehicleZ);

    int3 gridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(vehiclePos);

    for(int z=-2;z<=2;z++)
    {
        for(int y=-1;y<=1;y++)
        {
            for(int x=-2;x<=2;x++)
            {
                const int3 neighbourGridCellCoordinate = gridCellCoordinate + make_int3(x,y,z);
                const int neighbourGridCellIndex = parametersPathPlanner.grid.getSafeCellHash(neighbourGridCellCoordinate);
                float3 cellCenter = parametersPathPlanner.grid.getCellCenter(neighbourGridCellCoordinate);
                printf("clearOccupancyGridAboveVehiclePositionD(): clearing cell at %.2f / %.2f / %.2f\n", cellCenter.x, cellCenter.y, cellCenter.z);
                gridValues[neighbourGridCellIndex] = 0;
            }
        }
    }
}


void clearOccupancyGridAboveVehiclePosition(
        unsigned char* gridValues,
        float vehicleX,
        float vehicleY,
        float vehicleZ,
        hipStream_t *stream)
{
    clearOccupancyGridAboveVehiclePositionD<<< 1, 1, 0, *stream>>>(gridValues, vehicleX, vehicleY, vehicleZ);
    cudaCheckSuccess("clearOccupancyGridAboveVehiclePosition");
}

__global__ void moveWayPointsToSafetyD(unsigned char* gridValues, float4* deviceWaypoints, unsigned int numberOfWayPoints)
{
    uint wptIndex = getThreadIndex1D();
    if(wptIndex >= numberOfWayPoints) return;

    float4 waypoint = deviceWaypoints[wptIndex];

    int3 gridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(make_float3(waypoint));

    int searchOrderHorizontal[3];
    searchOrderHorizontal[0] = 0;
    searchOrderHorizontal[1] = -1;
    searchOrderHorizontal[2] = +1;

    bool freeCellFound = false;
    if(wptIndex == 0) printf("cell height is %.2f meters\n", parametersPathPlanner.grid.getCellSize().y);

    // With a scanner range of 15m, how many cells should we search upwards of the waypoint candidate?
    unsigned int maxNumberOfGridCellsToGoUp = 15.0 / parametersPathPlanner.grid.getCellSize().y;
    printf("waypoint %d at %.2f/%.2f/%.2f will search %d cells up.\n", wptIndex, waypoint.x, waypoint.y, waypoint.z, maxNumberOfGridCellsToGoUp);

    for(int z=0;z<3 && !freeCellFound;z++)
    {
        for(int x=0;x<3 && !freeCellFound;x++)
        {
            for(int y=2;y<maxNumberOfGridCellsToGoUp && !freeCellFound;y++)
            {
                const int3 neighbourGridCellCoordinate = gridCellCoordinate + make_int3(searchOrderHorizontal[x],y,searchOrderHorizontal[z]);
                const int neighbourGridCellIndex = parametersPathPlanner.grid.getSafeCellHash(neighbourGridCellCoordinate);

                if(gridValues[neighbourGridCellIndex] == 0)
                {
                    freeCellFound = true;
                    float3 cellCenter = parametersPathPlanner.grid.getCellCenter(neighbourGridCellCoordinate);
                    deviceWaypoints[wptIndex] = make_float4(cellCenter, waypoint.w);
                    printf("waypoint %d found free neighbor at %.2f/%.2f/%.2f.\n", wptIndex, cellCenter.x, cellCenter.y, cellCenter.z);
                }
            }
        }
    }

    // The waypoint is unusable, remove it!
    if(!freeCellFound)
    {
        printf("waypoint %d found no free neighbor.\n");
        deviceWaypoints[wptIndex] = make_float4(0.0);
    }
}

void moveWayPointsToSafetyGpu(
        unsigned char*  gridOccupancy,
        float*          mDeviceWaypoints,
        unsigned int    numberOfWayPoints,
        hipStream_t*   stream)
{
    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numberOfWayPoints, 64, numBlocks, numThreads);

    moveWayPointsToSafetyD<<< numBlocks, numThreads, 0, *stream>>>(gridOccupancy, (float4*)mDeviceWaypoints, numberOfWayPoints);
    cudaCheckSuccess("moveWayPointsToSafetyGpu");
}

void fillOccupancyGrid(unsigned char* gridValues, float* colliderPos, unsigned int numColliders, unsigned int numCells, hipStream_t *stream)
{
    if(numColliders == 0) return;

    // set all cells to empty
    hipMemset(gridValues, 0, numCells * sizeof(unsigned char));

    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numColliders, 64, numBlocks, numThreads);

    printf("fillOccupancyGrid(): using %d colliders at %p to fill occupancy grid with %d cells at %p.\n",
           numColliders, colliderPos, numCells, gridValues);

    fillOccupancyGridD<<< numBlocks, numThreads, 0, *stream>>>(gridValues, (float4*)colliderPos, numColliders);
    cudaCheckSuccess("fillOccupancyGrid");

    printf("fillOccupancyGrid(): done.\n");
}

void dilateOccupancyGrid(unsigned char* gridValues, unsigned int numCells, hipStream_t *stream)
{
    printf("dilateOccupancyGrid(): dilating %d cells.\n", numCells);

    if(numCells == 0) return;

    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numCells, 64, numBlocks, numThreads);

    dilateOccupancyGridD<<< numBlocks, numThreads, 0, *stream>>>(gridValues, numCells);
    cudaCheckSuccess("dilateOccupancyGridD");

    printf("dilateOccupancyGrid(): done.\n");
}

__device__
int bound(int min, int value, int max)
{
    if(value < min)
        return min;
    else if(value > max)
        return max;
    else
        return value;
}

__global__ void markStartCellD(u_int8_t* gridValues)
{
    int3 cellCoordinateStart = parametersPathPlanner.grid.getCellCoordinate(parametersPathPlanner.start);
    int cellIndexStart = parametersPathPlanner.grid.getSafeCellHash(cellCoordinateStart);

    if(cellIndexStart > 0)
    {
        printf("markStartCellD(): setting start cell %d to 1\n", cellIndexStart);
        gridValues[cellIndexStart] = 1;
    }
    else
    {
        printf("markStartCellD(): start cell %.1f/%.1f/%.1f is outside grid!\n", parametersPathPlanner.start.x, parametersPathPlanner.start.y, parametersPathPlanner.start.z);
    }
}

__global__
void growGridD(u_int8_t* gridValues, Grid subGrid)
{
    uint subGridCellHash = getThreadIndex1D();
    if(subGridCellHash >= subGrid.getCellCount()) return;

    float3 subGridCellCenter = subGrid.getCellCenter(subGrid.getCellCoordinate(subGridCellHash));
    int3 superGridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(subGridCellCenter);
    unsigned int superGridCellHash = parametersPathPlanner.grid.getCellHash(superGridCellCoordinate);

    u_int8_t lowestNonNullNeighbor = 254; // thats a dilated cell's value
    u_int8_t ownValue = gridValues[superGridCellHash];

    if(ownValue == 0)
    {
        // Check all neighbors for the lowest value d != 0,254,255 and put d++ into our own cell.
        for(int z=-1;z<=1;z++)
        {
            for(int y=-1;y<=1;y++)
            {
                for(int x=-1;x<=1;x++)
                {
                    // don't look into our own cell for neighboring values!
                    if(x == 0 && y == 0 && z == 0)
                    {
                        //printf("will not check myself for neighbors.\n");
                        continue;
                    }

                    const int3 neighbourGridCellCoordinate = superGridCellCoordinate + make_int3(x,y,z);

                    // Border-cells might ask for neighbors outside of the grid.
                    if(parametersPathPlanner.grid.isCellInGrid(neighbourGridCellCoordinate))
                    {
                        const int neighbourGridCellIndex = parametersPathPlanner.grid.getCellHash(neighbourGridCellCoordinate);
                        const u_int8_t neighborValue = gridValues[neighbourGridCellIndex];

                        // Find the lowest neighbor that is neither 0 nor 255
                        if(neighborValue < lowestNonNullNeighbor && neighborValue != 0)
                            lowestNonNullNeighbor = neighborValue;
                    }
                    else
                    {
                        // @subGrid should be clamped to the super grid, so this happens only when checking the non-existing neighbors of border cells
                        /*printf("bug, neighborgridcellindex is %d, super-coord was %d/%d/%d, neighbor-coord was %d/%d/%d\n",
                               neighbourGridCellIndex,
                               superGridCellCoordinate.x,
                               superGridCellCoordinate.y,
                               superGridCellCoordinate.z,
                               neighbourGridCellCoordinate.x,
                               neighbourGridCellCoordinate.y,
                               neighbourGridCellCoordinate.z);*/
                    }
                }
            }
        }

        // Write our cell's value. A cell first contains a 0, then the neighborCellValue+1. Once it does
        // contain a value, it will never change. We're only interested in replacing the value with lower
        // numbers, but since the values spread like a wave, that'll never happen.
        if(lowestNonNullNeighbor < 254/* && ownValue == 0*/)
        {
            /*printf("found value %d in neighbor, setting sub-cell %d / super-cell %d (%d/%d/%d) from %d to %d\n",
                   lowestNonNullNeighbor,
                   subGridCellHash,
                   superGridCellHash,
                   superGridCellCoordinate.x, superGridCellCoordinate.y, superGridCellCoordinate.z,
                   ownValue,
                   lowestNonNullNeighbor + 1);*/

            gridValues[superGridCellHash] = lowestNonNullNeighbor + 1;
        }
        else
        {
            /*printf("failed to find an interesting neighbor for sub-grid-cell %d, super-grid-cell %d (%3d/%3d/%3d) with value %d\n",
                   subGridCellHash,
                   superGridCellHash,
                   superGridCellCoordinate.x,
                   superGridCellCoordinate.y,
                   superGridCellCoordinate.z,
                   ownValue);*/
        }
    }
    else
    {
        /*printf("sub-grid-cell %d, super-grid-cell %d (%3d/%3d/%3d) already has value %d\n",
               subGridCellHash,
               superGridCellHash,
               superGridCellCoordinate.x,
               superGridCellCoordinate.y,
               superGridCellCoordinate.z,
               ownValue);*/
    }
}

void markStartCell(unsigned char* gridValues, hipStream_t *stream)
{
    // set the cell containing "start" to 1!
    markStartCellD<<<1, 1, 0, *stream>>>(gridValues);
    cudaCheckSuccess("markStartCellD");
}

void growGrid(unsigned char* gridValues, ParametersPathPlanner* parameters, hipStream_t *stream)
{
    uint numThreads, numBlocks;

    const int3 cellCoordinateStart = parameters->grid.getCellCoordinate(parameters->start);


    const unsigned int longestSideCellCount = parameters->grid.getLongestSideCellCount();

    int3 iterationCellMin, iterationCellMax, lastCellMin, lastCellMax;

    for(unsigned int i=1;i<longestSideCellCount;i++)
    {
        iterationCellMin = parameters->grid.clampCellCoordinate(cellCoordinateStart + make_int3(-i, -i, -i));
        iterationCellMax = parameters->grid.clampCellCoordinate(cellCoordinateStart + make_int3(+i, +i, +i));

        if(iterationCellMin == lastCellMin && iterationCellMax == lastCellMax)
        {
            // cell coordinates haven't changed, so we have grown the whole grid.
            break;
        }
        else
        {
            lastCellMin = iterationCellMin;
            lastCellMax = iterationCellMax;
        }

        Grid iterationGrid;
        iterationGrid.cells.x = iterationCellMax.x - iterationCellMin.x + 1;
        iterationGrid.cells.y = iterationCellMax.y - iterationCellMin.y + 1;
        iterationGrid.cells.z = iterationCellMax.z - iterationCellMin.z + 1;
        float3 superGridCellSize = parameters->grid.getCellSize();
        iterationGrid.worldMin = parameters->grid.getCellCenter(iterationCellMin) - superGridCellSize/2;
        iterationGrid.worldMax = parameters->grid.getCellCenter(iterationCellMax) + superGridCellSize/2;

//        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(growingGrid), iterationGrid, sizeof(Grid)));

        computeExecutionKernelGrid(iterationGrid.getCellCount(), 64, numBlocks, numThreads);
        //printf("growGrid(): growing grid in %d cells.\n", iterationGrid.getCellCount());
        growGridD<<< numBlocks, numThreads, 0, *stream>>>(gridValues, iterationGrid);
        cudaCheckSuccess("growGridD");
    }
}

__global__
void checkGoalCellD(unsigned char* gridValues, unsigned int numCells, unsigned int searchRange, unsigned int *status)
{
    int3 goalGridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(parametersPathPlanner.goal);
    int goalGridCellOffset = parametersPathPlanner.grid.getSafeCellHash(goalGridCellCoordinate);

    uint valueInGoalCell = gridValues[goalGridCellOffset];

    printf("checkGoalCellD(): value in goal cell at %.2f/%.2f/%.2f is %d.\n",
           parametersPathPlanner.goal.x,
           parametersPathPlanner.goal.y,
           parametersPathPlanner.goal.z,
           valueInGoalCell);

    if(valueInGoalCell < 254)
    {
        return;
    }
    else
    {
        // Cell is occupied or dilated-occupied! Try to find an empty neighbor!

        // With searchRange = 3, create an array {1,-1,2,-2,3,-3}
        float *neighborsSearchOrder = new float[searchRange * 2];
        for(int i=1;i<=searchRange;i++)
        {
            neighborsSearchOrder[2*i-2] = i;
            neighborsSearchOrder[2*i-1] = -i;
        }

        //for(...)

        delete neighborsSearchOrder;
    }
}


// This method checks whether the goal cell is occupied. If so, it tries
// to find a free neighboring cell that can be used instead.
GoalCellStatus checkGoalCell(unsigned char* gridValues, unsigned int numCells, unsigned int searchRange, hipStream_t *stream)
{
    if(numCells == 0) return GoalCellBlocked;

    u_int32_t* statusDevice = 0;
    cudaSafeCall(hipMalloc((void**)statusDevice, sizeof(u_int32_t)));

    checkGoalCellD<<< 1, 1, 0, *stream>>>(gridValues, numCells, searchRange, statusDevice);
    cudaCheckSuccess("checkGoalCell");

    u_int32_t statusHost;
    cudaSafeCall(hipMemcpy(&statusHost, statusDevice, sizeof(u_int32_t), hipMemcpyDeviceToHost));

    if(statusHost == 0)
    {
        return GoalCellFree;
    }
    else if(statusHost == 1)
    {
        return GoalCellMoved;
    }
    else
    {
        return GoalCellBlocked;
    }
}



__global__
void retrievePathD(unsigned char* gridValues, float4* waypoints)
{
    int3 gridCellCoordinateGoal = parametersPathPlanner.grid.getCellCoordinate(parametersPathPlanner.goal);
    int gridCellOffsetGoal = parametersPathPlanner.grid.getSafeCellHash(gridCellCoordinateGoal);

    int3 gridCellCoordinateStart = parametersPathPlanner.grid.getCellCoordinate(parametersPathPlanner.start);

    uint valueInGoalCell = gridValues[gridCellOffsetGoal];
    printf("retrievePathD(): value in goal cell at %.2f/%.2f/%.2f is %d.\n",
           parametersPathPlanner.goal.x,
           parametersPathPlanner.goal.y,
           parametersPathPlanner.goal.z,
           valueInGoalCell);

    if(valueInGoalCell == 0)
    {
        // Tell the caller we failed to find a valid path by setting the first waypoint to all-zero.
        waypoints[0] = make_float4(0.0, 0.0, 0.0, 0.0);
    }
    else if(valueInGoalCell == 255 || valueInGoalCell == 254)
    {
        // Tell the caller we failed to find a valid path because of an occupied target cell.
        waypoints[0] = make_float4(0.0, 0.0, 0.0, 1.0);
    }
    else
    {
        // Use this ONE thread to collect all the waypoints. The first float4 will contain
        // the number of waypoints including start and goal. The next float4s will be those
        // waypoints. Add 0.1 so we can cast to int without losing something.
        waypoints[0] = make_float4(valueInGoalCell + 0.1);

        // Set the last waypoint, which equals the goal position
        waypoints[valueInGoalCell] = make_float4(parametersPathPlanner.goal);

        // Now traverse from goal back to start and save the world positions in waypoints
        uint stepsToStartCell = valueInGoalCell;
        int3 cellCoordinate = gridCellCoordinateGoal;

        // Saves the direction/offset that we step to get to the next cell.
        int3 lastCellOffset;

        do
        {
            // We are at cellCoordinate and found a value of distance. Now check all neighbors
            // until we find one with a smaller value. Thats the path backwards towards the goal.

            bool foundNextCellTowardsTarget = false;

            if(!foundNextCellTowardsTarget)
            {
                // Paths found using the three nested loops below often look strange, because we search
                // in certain directions first. To prevent this, we first search the cell towards the
                // direction of the goal...

                int3 cellOffset = make_int3(
                            cudaBound(-1, gridCellCoordinateStart.x - cellCoordinate.x, 1),
                            cudaBound(-1, gridCellCoordinateStart.y - cellCoordinate.y, 1),
                            cudaBound(-1, gridCellCoordinateStart.z - cellCoordinate.z, 1));

                int3 neighbourCellCoordinate = cellCoordinate + cellOffset;
                int neighbourCellIndex = parametersPathPlanner.grid.getSafeCellHash(neighbourCellCoordinate);
                if(neighbourCellIndex < 0) printf("bug!!!");

                u_int8_t neighborValue = gridValues[neighbourCellIndex];

                if(neighborValue < stepsToStartCell)
                {
                    printf("retrievePathD(): found next cell by stepping towards target: %d/%d/%d\n", cellOffset.x, cellOffset.y, cellOffset.z);

                    // We found a neighbor with a smaller distance. Use it!
                    cellCoordinate = neighbourCellCoordinate;

                    // Save this step for the next iteration!
                    lastCellOffset = cellOffset;

                    // Append our current cell's position to the waypoint list.
                    float3 cellCenter = parametersPathPlanner.grid.getCellCenter(cellCoordinate);
                    //printf("retrievePathD(): found next cell towards start at %.2f/%.2f/%.2f%d.\n", cellCenter.x, cellCenter.y, cellCenter.z);
                    waypoints[neighborValue] = make_float4(cellCenter);

                    // Escape those 3 for-loops to continue searching from this next cell.
                    foundNextCellTowardsTarget = true;

                    // Update distance to start-position, should be a simple decrement.
                    stepsToStartCell = neighborValue;
                }
            }

            if(!foundNextCellTowardsTarget)
            {
                // Ok, the direct step didn't work.
                // Define search order. First try to repeat the last step. If that fails, at least try to keep the height.
                int searchOrderX[3];
                if(lastCellOffset.x == 0)
                {
                    searchOrderX[0] = lastCellOffset.x;
                    searchOrderX[1] = -1;
                    searchOrderX[2] = +1;
                }
                else
                {
                    searchOrderX[0] = lastCellOffset.x;
                    searchOrderX[1] = +0;
                    searchOrderX[2] = -lastCellOffset.x;
                }

                int searchOrderY[3];
                if(lastCellOffset.y == 0)
                {
                    searchOrderY[0] = lastCellOffset.y;
                    searchOrderY[1] = -1;
                    searchOrderY[2] = +1;
                }
                else
                {
                    searchOrderY[0] = lastCellOffset.y;
                    searchOrderY[1] = +0;
                    searchOrderY[2] = -lastCellOffset.y;
                }

                int searchOrderZ[3];
                if(lastCellOffset.z == 0)
                {
                    searchOrderZ[0] = lastCellOffset.z;
                    searchOrderZ[1] = -1;
                    searchOrderZ[2] = +1;}
                else
                {
                    searchOrderZ[0] = lastCellOffset.z;
                    searchOrderZ[1] = +0;
                    searchOrderZ[2] = -lastCellOffset.z;
                }

                /*searchOrderX[0] = 0;
                searchOrderX[1] = 1;
                searchOrderX[2] = -1;

                searchOrderY[0] = 0;
                searchOrderY[1] = 1;
                searchOrderY[2] = -1;

                searchOrderZ[0] = 0;
                searchOrderZ[1] = 1;
                searchOrderZ[2] = -1;*/

                // now search the neighbors in the given order.
                for(int z=0; z<3 && !foundNextCellTowardsTarget; z++)
                {
                    for(int y=0; y<3 && !foundNextCellTowardsTarget; y++) // check lower paths first
                    {
                        for(int x=0; x<3 && !foundNextCellTowardsTarget; x++)
                        {
                            int3 cellOffset = make_int3(searchOrderX[x], searchOrderY[y], searchOrderZ[z]);
                            int3 neighbourCellCoordinate = cellCoordinate + cellOffset;

                            int neighbourCellIndex = parametersPathPlanner.grid.getSafeCellHash(neighbourCellCoordinate);

                            if(neighbourCellIndex >= 0)
                            {
                                u_int8_t neighborValue = gridValues[neighbourCellIndex];

                                if(neighborValue < stepsToStartCell)
                                {
                                    if(x+y+z == 0)
                                        printf("retrievePathD(): x%d y%d z%d - found next cell by repeating last step: %d/%d/%d\n", x,y,z, cellOffset.x, cellOffset.y, cellOffset.z);
                                    else
                                        printf("retrievePathD(): x%d y%d z%d - found next cell using all neighbors: %d/%d/%d\n", x,y,z, cellOffset.x, cellOffset.y, cellOffset.z);

                                    // We found a neighbor with a smaller distance. Use it!
                                    cellCoordinate = neighbourCellCoordinate;

                                    lastCellOffset = cellOffset;

                                    // Append our current cell's position to the waypoint list.
                                    float3 cellCenter = parametersPathPlanner.grid.getCellCenter(cellCoordinate);
                                    //printf("retrievePathD(): found next cell towards start at %.2f/%.2f/%.2f%d.\n", cellCenter.x, cellCenter.y, cellCenter.z);

                                    // The w-component doesn't matter here, so set to zero. Later on, the w-component
                                    // will be set to 1 if it turns out that the waypoint is in a now-occupied cell.
                                    waypoints[neighborValue] = make_float4(cellCenter, 0.0);

                                    // Escape those 3 for-loops to continue searching from this next cell.
                                    foundNextCellTowardsTarget = true;

                                    // Update distance to start-position, should be a simple decrement.
                                    stepsToStartCell = neighborValue;
                                }
                            }
                        }
                    }
                }
            }
        }
        while(stepsToStartCell > 1);

        // waypoints[1] was filled above with the cell-center. But we want it to be the start-position, which
        // - although contained in the cell - is probably not exactly its center.
        waypoints[1] = make_float4(parametersPathPlanner.start);
    }
}


void retrievePath(unsigned char* gridValues, float *waypoints, hipStream_t *stream)
{
    retrievePathD<<< 1, 1, 0, *stream>>>(gridValues, (float4*)waypoints);
    cudaCheckSuccess("retrievePathD");
}

__global__ void testWayPointCellOccupancyD(unsigned char*  gridValues, float4* upcomingWayPoints, unsigned int numberOfWayPoints)
{
    uint waypointIndex = getThreadIndex1D();
    if(waypointIndex >= numberOfWayPoints) return;

    float4 waypoint = upcomingWayPoints[waypointIndex];

    const int3 gridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(make_float3(waypoint.x, waypoint.y, waypoint.z));
    const int gridCellHash = parametersPathPlanner.grid.getSafeCellHash(gridCellCoordinate);

    if(gridCellHash < 0 || gridCellHash > parametersPathPlanner.grid.getCellCount())
        printf("testWayPointCellOccupancyD(): bug, waypoint %d is supposedly at %.2f/%.2f/%.2f/%.2f in cell hash %d\n",
               waypointIndex, waypoint.x, waypoint.y, waypoint.z, waypoint.w, gridCellHash);

    if(gridValues[gridCellHash] > 253)
    {
        waypoint.w = 1.0;
        upcomingWayPoints[waypointIndex] = waypoint;
    }
}

void testWayPointCellOccupancy(unsigned char*  gridValues, float* upcomingWayPoints, unsigned int numberOfWayPoints, hipStream_t *stream)
{
    // We take the grid values and the float4-waypoints (with the first element defining the waypoint-count) and set conflicting

    // The number of waypoints available is in upcomingWayPoints[0], but thats in device memory space. So,
    // just start a sufficient number of threads and let the superfuous ones hang out for a while.
    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numberOfWayPoints, 64, numBlocks, numThreads);

    testWayPointCellOccupancyD<<< numBlocks, numThreads, 0, *stream>>>(gridValues, (float4*)upcomingWayPoints, numberOfWayPoints);
    cudaCheckSuccess("fillOccupancyGrid");
}
