#include "hip/hip_runtime.h"
// Fix for gcc 4.7
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include "grid.cuh"
#include "cudahelper.cuh"
#include "hip/hip_vector_types.h"

#include "parameterspathplanner.cuh"
#include "pathplanner.cuh"

// only for printf debugging
#include <stdlib.h>
#include <stdio.h>

// simulation parameters in constant memory
__constant__ ParametersPathPlanner parametersPathPlanner;
__constant__ Grid growingGrid;

void copyParametersToGpu(ParametersPathPlanner *hostParams)
{
    // Copy parameters to constant memory.
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(parametersPathPlanner), hostParams, sizeof(ParametersPathPlanner)));
}

__global__
void fillOccupancyGridD(u_int8_t* gridValues, const float4* colliderPos, unsigned int numColliders, unsigned int numCells)
{
    uint colliderIndex = getThreadIndex1D();

    if(colliderIndex >= numColliders) return;

    const float4 particlePosition = colliderPos[colliderIndex];

    if(parametersPathPlanner.grid.isPositionInGrid(particlePosition))
    {
        // get grid-cell of particle
        int3 particleGridCell = parametersPathPlanner.grid.getCellCoordinate(make_float3(particlePosition));

        // The cell-hash IS the offset in memory, as cells are adressed linearly
        int cellHash = parametersPathPlanner.grid.getCellHash(particleGridCell);

        if(cellHash >= 0 && cellHash < numCells)
        {
            gridValues[cellHash] = 255;
        }
        else
        {
            printf("ERROR, position was supposed to be in grid! We have %d cells and want to write to cell %d.\n\n\n", numCells, cellHash);
        }
    }
}

__global__
void dilateOccupancyGridD(u_int8_t* gridValues, unsigned int numCells)
{
    // Dilate the occupied cells for additional safety. This also allows expanding routes diagonally
    // later-on, as its ok to pass diagonally between occupied cells after dilation
    uint cellIndex = getThreadIndex1D();
    if(cellIndex >= numCells) return;

    u_int8_t ownValue = gridValues[cellIndex];
    if(ownValue == 255) return;

    int3 threadGridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(cellIndex);

    for(int z=-1;z<=1;z++)
    {
        for(int y=-1;y<=1;y++)
        {
            for(int x=-1;x<=1;x++)
            {
                const int3 neighbourGridCellCoordinate = threadGridCellCoordinate + make_int3(x,y,z);
                //if(cellIndex == 0) printf("cellIndex 0, coord 0/0/0 neighbor %d/%d/%d\n", x, y, z);

                if(parametersPathPlanner.grid.isCellInGrid(neighbourGridCellCoordinate))
                {
                    const int neighbourGridCellIndex = parametersPathPlanner.grid.getSafeCellHash(neighbourGridCellCoordinate);
                    // Because CUDA works using thread-batches, we cannot just load all cells, then compute and then store all cells.
                    // Using batches would mean we would dilate a part of the grid, then load the neighboring part and dilate the
                    // dilation again, making almost all of the grid become occupied.
                    // For this reason, we say that 255 is occupied and 254 is dilated-occupied. This way, we don't need two grids. Hah!
                    if(gridValues[neighbourGridCellIndex] == 255)
                    {
                        gridValues[cellIndex] = 254;
                        return;
                    }
                }
            }
        }
    }
}

__global__ void clearOccupancyGridAboveVehiclePositionD(
        unsigned char* gridValues,
        float vehicleX,
        float vehicleY,
        float vehicleZ)
{
    float3 vehiclePos = make_float3(vehicleX, vehicleY, vehicleZ);

    int3 gridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(vehiclePos);

    for(int z=-2;z<=2;z++)
    {
        for(int y=-1;y<=1;y++)
        {
            for(int x=-2;x<=2;x++)
            {
                const int3 neighbourGridCellCoordinate = gridCellCoordinate + make_int3(x,y,z);
                const int neighbourGridCellIndex = parametersPathPlanner.grid.getSafeCellHash(neighbourGridCellCoordinate);
                float3 cellCenter = parametersPathPlanner.grid.getCellCenter(neighbourGridCellCoordinate);
                printf("clearOccupancyGridAboveVehiclePositionD(): clearing cell at %.2f / %.2f / %.2f\n", cellCenter.x, cellCenter.y, cellCenter.z);
                gridValues[neighbourGridCellIndex] = 0;
            }
        }
    }
}


void clearOccupancyGridAboveVehiclePosition(
        unsigned char* gridValues,
        float vehicleX,
        float vehicleY,
        float vehicleZ,
        hipStream_t *stream)
{
    clearOccupancyGridAboveVehiclePositionD<<< 1, 1, 0, *stream>>>(gridValues, vehicleX, vehicleY, vehicleZ);
    cudaCheckSuccess("clearOccupancyGridAboveVehiclePosition");
}

__global__ void moveWayPointsToSafetyD(unsigned char* gridValues, float4* deviceWaypoints, unsigned int numberOfWayPoints)
{
    uint wptIndex = getThreadIndex1D();
    if(wptIndex >= numberOfWayPoints) return;

    float4 waypoint = deviceWaypoints[wptIndex];

    if(!parametersPathPlanner.grid.isPositionInGrid(waypoint))
    {
        printf("error, waypoint %d is not even in the grid!\n", wptIndex);
        deviceWaypoints[wptIndex] = make_float4(0.0);
        return;
    }

    int3 gridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(make_float3(waypoint));

    if(!parametersPathPlanner.grid.isCellInGrid(gridCellCoordinate))
    {
        printf("moveWayPointsToSafetyD: error, this doesn't make sense at all!");
        return;
    }

    int searchOrderHorizontal[3];
    searchOrderHorizontal[0] = 0;
    searchOrderHorizontal[1] = -1;
    searchOrderHorizontal[2] = +1;

    bool freeCellFound = false;
    if(wptIndex == 0) printf("cell height is %.2f meters\n", parametersPathPlanner.grid.getCellSize().y);

    // With a scanner range of 15m, how many cells should we search upwards of the waypoint candidate?
    unsigned int maxNumberOfGridCellsToGoUp = 15.0 / parametersPathPlanner.grid.getCellSize().y;
    printf("waypoint %d at %.2f/%.2f/%.2f will search up to %d cells up.\n", wptIndex, waypoint.x, waypoint.y, waypoint.z, maxNumberOfGridCellsToGoUp);

    for(int z=0;z<3 && !freeCellFound;z++)
    {
        for(int x=0;x<3 && !freeCellFound;x++)
        {
            for(int y=0;y<maxNumberOfGridCellsToGoUp && !freeCellFound;y++)
            {
                const int3 neighbourGridCellCoordinate = gridCellCoordinate + make_int3(searchOrderHorizontal[x],y,searchOrderHorizontal[z]);
                if(parametersPathPlanner.grid.isCellInGrid(neighbourGridCellCoordinate))
                {
                    const int neighbourGridCellIndex = parametersPathPlanner.grid.getSafeCellHash(neighbourGridCellCoordinate);

                    if(gridValues[neighbourGridCellIndex] == 0)
                    {
                        freeCellFound = true;
                        float3 cellCenter = parametersPathPlanner.grid.getCellCenter(neighbourGridCellCoordinate);
                        deviceWaypoints[wptIndex] = make_float4(cellCenter, waypoint.w);
                        printf("waypoint %d found free neighbor at %.2f/%.2f/%.2f.\n", wptIndex, cellCenter.x, cellCenter.y, cellCenter.z);
                    }
                }
            }
        }
    }

    // The waypoint is unusable, remove it!
    if(!freeCellFound)
    {
        printf("waypoint %d found no free neighbor.\n", wptIndex);
        deviceWaypoints[wptIndex] = make_float4(0.0);
    }
}

// Will move the waypoints to cells that are free in gridOccupancy. If the w-component is untouched (and non-zero),
// it was possible to move them to free zones. Waypoints with w-component of zero could not find a free neighboring cell.
void moveWayPointsToSafetyGpu(
        unsigned char*  gridOccupancy,
        float*          mDeviceWaypoints,
        unsigned int    numberOfWayPoints,
        hipStream_t*   stream)
{
    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numberOfWayPoints, 64, numBlocks, numThreads);

    moveWayPointsToSafetyD<<< numBlocks, numThreads, 0, *stream>>>(gridOccupancy, (float4*)mDeviceWaypoints, numberOfWayPoints);
    cudaCheckSuccess("moveWayPointsToSafetyGpu");
}

void fillOccupancyGrid(unsigned char* gridValues, const float* colliderPos, unsigned int numColliders, unsigned int numCells, hipStream_t *stream)
{
    if(numColliders == 0) return;

    // set all cells to empty
    cudaSafeCall(hipMemset(gridValues, 0, numCells * sizeof(unsigned char)));

    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numColliders, 64, numBlocks, numThreads);

    printf("fillOccupancyGrid(): using %d colliders at %p to fill occupancy grid with %d cells at %p.\n", numColliders, colliderPos, numCells, gridValues);

    fillOccupancyGridD<<< numBlocks, numThreads, 0, *stream>>>(gridValues, (float4*)colliderPos, numColliders, numCells);
    cudaCheckSuccess("fillOccupancyGrid");

    printf("fillOccupancyGrid(): done.\n");
}

void dilateOccupancyGrid(unsigned char* gridValues, unsigned int numCells, hipStream_t *stream)
{
    printf("dilateOccupancyGrid(): dilating %d cells.\n", numCells);

    if(numCells == 0) return;

    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numCells, 64, numBlocks, numThreads);

    cudaCheckSuccess("dilateOccupancyGridDBefore");
    dilateOccupancyGridD<<< numBlocks, numThreads, 0, *stream>>>(gridValues, numCells);
    cudaCheckSuccess("dilateOccupancyGridDAfter");

    printf("dilateOccupancyGrid(): done.\n");
}

__device__
int bound(int min, int value, int max)
{
    if(value < min)
        return min;
    else if(value > max)
        return max;
    else
        return value;
}

__global__ void markStartCellD(u_int8_t* gridValues)
{
    int3 cellCoordinateStart = parametersPathPlanner.grid.getCellCoordinate(parametersPathPlanner.start);
    int cellIndexStart = parametersPathPlanner.grid.getSafeCellHash(cellCoordinateStart);

    if(cellIndexStart > 0)
    {
        printf("markStartCellD(): setting start cell %d to 1\n", cellIndexStart);
        gridValues[cellIndexStart] = 1;
    }
    else
    {
        printf("markStartCellD(): start cell %.1f/%.1f/%.1f is outside grid!\n", parametersPathPlanner.start.x, parametersPathPlanner.start.y, parametersPathPlanner.start.z);
    }
}

__global__
void growGridD(u_int8_t* gridValues, Grid subGrid)
{
    uint subGridCellHash = getThreadIndex1D();
    if(subGridCellHash >= subGrid.getCellCount()) return;

    float3 subGridCellCenter = subGrid.getCellCenter(subGrid.getCellCoordinate(subGridCellHash));
    int3 superGridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(subGridCellCenter);
    unsigned int superGridCellHash = parametersPathPlanner.grid.getCellHash(superGridCellCoordinate);

    u_int8_t lowestNonNullNeighbor = 254; // thats a dilated cell's value
    u_int8_t ownValue = gridValues[superGridCellHash];

    if(ownValue == 0)
    {
        // Check all neighbors for the lowest value d != 0,254,255 and put d++ into our own cell.
        for(int z=-1;z<=1;z++)
        {
            for(int y=-1;y<=1;y++)
            {
                for(int x=-1;x<=1;x++)
                {
                    // don't look into our own cell for neighboring values!
                    if(x == 0 && y == 0 && z == 0)
                    {
                        //printf("will not check myself for neighbors.\n");
                        continue;
                    }

                    const int3 neighbourGridCellCoordinate = superGridCellCoordinate + make_int3(x,y,z);

                    // Border-cells might ask for neighbors outside of the grid.
                    if(parametersPathPlanner.grid.isCellInGrid(neighbourGridCellCoordinate))
                    {
                        const int neighbourGridCellIndex = parametersPathPlanner.grid.getCellHash(neighbourGridCellCoordinate);
                        const u_int8_t neighborValue = gridValues[neighbourGridCellIndex];

                        // Find the lowest neighbor that is neither 0 nor 255
                        if(neighborValue < lowestNonNullNeighbor && neighborValue != 0)
                            lowestNonNullNeighbor = neighborValue;
                    }
                    else
                    {
                        // @subGrid should be clamped to the super grid, so this happens only when checking the non-existing neighbors of border cells
                        /*printf("bug, neighborgridcellindex is %d, super-coord was %d/%d/%d, neighbor-coord was %d/%d/%d\n",
                               neighbourGridCellIndex,
                               superGridCellCoordinate.x,
                               superGridCellCoordinate.y,
                               superGridCellCoordinate.z,
                               neighbourGridCellCoordinate.x,
                               neighbourGridCellCoordinate.y,
                               neighbourGridCellCoordinate.z);*/
                    }
                }
            }
        }

        // Write our cell's value. A cell first contains a 0, then the neighborCellValue+1. Once it does
        // contain a value, it will never change. We're only interested in replacing the value with lower
        // numbers, but since the values spread like a wave, that'll never happen.
        if(lowestNonNullNeighbor < 254/* && ownValue == 0*/)
        {
            /*printf("found value %d in neighbor, setting sub-cell %d / super-cell %d (%d/%d/%d) from %d to %d\n",
                   lowestNonNullNeighbor,
                   subGridCellHash,
                   superGridCellHash,
                   superGridCellCoordinate.x, superGridCellCoordinate.y, superGridCellCoordinate.z,
                   ownValue,
                   lowestNonNullNeighbor + 1);*/

            gridValues[superGridCellHash] = lowestNonNullNeighbor + 1;
        }
        else
        {
            /*printf("failed to find an interesting neighbor for sub-grid-cell %d, super-grid-cell %d (%3d/%3d/%3d) with value %d\n",
                   subGridCellHash,
                   superGridCellHash,
                   superGridCellCoordinate.x,
                   superGridCellCoordinate.y,
                   superGridCellCoordinate.z,
                   ownValue);*/
        }
    }
    else
    {
        /*printf("sub-grid-cell %d, super-grid-cell %d (%3d/%3d/%3d) already has value %d\n",
               subGridCellHash,
               superGridCellHash,
               superGridCellCoordinate.x,
               superGridCellCoordinate.y,
               superGridCellCoordinate.z,
               ownValue);*/
    }
}

void markStartCell(unsigned char* gridValues, hipStream_t *stream)
{
    // set the cell containing "start" to 1!
    markStartCellD<<<1, 1, 0, *stream>>>(gridValues);
    cudaCheckSuccess("markStartCellD");
}

void growGrid(unsigned char* gridValues, ParametersPathPlanner* parameters, hipStream_t *stream)
{
    uint numThreads, numBlocks;

    const int3 cellCoordinateStart = parameters->grid.getCellCoordinate(parameters->start);


    const unsigned int longestSideCellCount = parameters->grid.getLongestSideCellCount();

    int3 iterationCellMin, iterationCellMax, lastCellMin, lastCellMax;

    for(unsigned int i=1;i<longestSideCellCount;i++)
    {
        iterationCellMin = parameters->grid.clampCellCoordinate(cellCoordinateStart + make_int3(-i, -i, -i));
        iterationCellMax = parameters->grid.clampCellCoordinate(cellCoordinateStart + make_int3(+i, +i, +i));

        if(iterationCellMin == lastCellMin && iterationCellMax == lastCellMax)
        {
            // cell coordinates haven't changed, so we have grown the whole grid.
            break;
        }
        else
        {
            lastCellMin = iterationCellMin;
            lastCellMax = iterationCellMax;
        }

        Grid iterationGrid;
        iterationGrid.cells.x = iterationCellMax.x - iterationCellMin.x + 1;
        iterationGrid.cells.y = iterationCellMax.y - iterationCellMin.y + 1;
        iterationGrid.cells.z = iterationCellMax.z - iterationCellMin.z + 1;
        float3 superGridCellSize = parameters->grid.getCellSize();
        iterationGrid.worldMin = parameters->grid.getCellCenter(iterationCellMin) - superGridCellSize/2;
        iterationGrid.worldMax = parameters->grid.getCellCenter(iterationCellMax) + superGridCellSize/2;

//        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(growingGrid), iterationGrid, sizeof(Grid)));

        computeExecutionKernelGrid(iterationGrid.getCellCount(), 64, numBlocks, numThreads);
        //printf("growGrid(): growing grid in %d cells.\n", iterationGrid.getCellCount());
        growGridD<<< numBlocks, numThreads, 0, *stream>>>(gridValues, iterationGrid);
        cudaCheckSuccess("growGridD");
    }
}

__global__
void checkGoalCellD(unsigned char* gridValues, unsigned int numCells, unsigned int searchRange, unsigned int *status)
{
    int3 goalGridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(parametersPathPlanner.goal);
    int goalGridCellOffset = parametersPathPlanner.grid.getSafeCellHash(goalGridCellCoordinate);

    uint valueInGoalCell = gridValues[goalGridCellOffset];

    printf("checkGoalCellD(): value in goal cell at %.2f/%.2f/%.2f is %d.\n",
           parametersPathPlanner.goal.x,
           parametersPathPlanner.goal.y,
           parametersPathPlanner.goal.z,
           valueInGoalCell);

    if(valueInGoalCell < 254)
    {
        return;
    }
    else
    {
        // Cell is occupied or dilated-occupied! Try to find an empty neighbor!

        // With searchRange = 3, create an array {1,-1,2,-2,3,-3}
        float *neighborsSearchOrder = new float[searchRange * 2];
        for(int i=1;i<=searchRange;i++)
        {
            neighborsSearchOrder[2*i-2] = i;
            neighborsSearchOrder[2*i-1] = -i;
        }

        //for(...)

        delete neighborsSearchOrder;
    }
}

// This method checks whether the goal cell is occupied. If so, it tries
// to find a free neighboring cell that can be used instead.
GoalCellStatus checkGoalCell(unsigned char* gridValues, unsigned int numCells, unsigned int searchRange, hipStream_t *stream)
{
    if(numCells == 0) return GoalCellBlocked;

    u_int32_t* statusDevice = 0;
    cudaSafeCall(hipMalloc((void**)statusDevice, sizeof(u_int32_t)));

    checkGoalCellD<<< 1, 1, 0, *stream>>>(gridValues, numCells, searchRange, statusDevice);
    cudaCheckSuccess("checkGoalCell");

    u_int32_t statusHost;
    cudaSafeCall(hipMemcpy(&statusHost, statusDevice, sizeof(u_int32_t), hipMemcpyDeviceToHost));

    if(statusHost == 0)
    {
        return GoalCellFree;
    }
    else if(statusHost == 1)
    {
        return GoalCellMoved;
    }
    else
    {
        return GoalCellBlocked;
    }
}

__global__
void retrievePathD(unsigned char* gridValues, float4* waypoints)
{
    int3 gridCellGoalCoordinate = parametersPathPlanner.grid.getCellCoordinate(parametersPathPlanner.goal);
    int gridCellGoalHash = parametersPathPlanner.grid.getSafeCellHash(gridCellGoalCoordinate);

    int3 gridCellCoordinateStart = parametersPathPlanner.grid.getCellCoordinate(parametersPathPlanner.start);

    uint valueInGoalCell = gridValues[gridCellGoalHash];
    printf("retrievePathD(): value in goal cell at %.2f/%.2f/%.2f is %d.\n",
           parametersPathPlanner.goal.x,
           parametersPathPlanner.goal.y,
           parametersPathPlanner.goal.z,
           valueInGoalCell);

    if(valueInGoalCell == 0)
    {
        // Tell the caller we failed to find a valid path by setting the first waypoint to all-zero.
        waypoints[0] = make_float4(0.0, 0.0, 0.0, 0.0);
    }
    else if(valueInGoalCell >= 254)
    {
        // Tell the caller we failed to find a valid path because of an occupied target cell.
        waypoints[0] = make_float4(0.0, 0.0, 0.0, 1.0);
    }
    else
    {
        // Use this ONE thread to collect all the waypoints. The first float4 will contain
        // the number of waypoints including start and goal. The next float4s will be those
        // waypoints. Add 0.1 so we can cast to int without losing something.
        waypoints[0] = make_float4(valueInGoalCell + 0.1);

        // Set the last waypoint, which equals the goal position
        waypoints[valueInGoalCell] = make_float4(parametersPathPlanner.goal);

        // Now traverse from goal back to start and save the world positions in waypoints
        uint stepsToStartCell = valueInGoalCell;
        int3 currentCellCoordinate = gridCellGoalCoordinate;

        // Saves the direction/offset that we step to get to the next cell.
        int3 lastTravelDirection;

        do
        {
            // We are at cellCoordinate and found a value of distance. Now check all neighbors
            // until we find one with a smaller value. That's the path backwards towards the goal.

            bool foundNextCellTowardsTarget = false;

            if(!foundNextCellTowardsTarget)
            {
                // Paths found using the three nested loops below often look strange, because we search
                // in certain directions first. To prevent this, we first search the cell towards the
                // direction of the goal...

                int3 travelDirectionDirect = make_int3(
                            cudaBound(-1, gridCellCoordinateStart.x - currentCellCoordinate.x, 1),
                            cudaBound(-1, gridCellCoordinateStart.y - currentCellCoordinate.y, 1),
                            cudaBound(-1, gridCellCoordinateStart.z - currentCellCoordinate.z, 1));

                int3 neighbourCellCoordinate = currentCellCoordinate + travelDirectionDirect;

                if(parametersPathPlanner.grid.isCellInGrid(neighbourCellCoordinate))
                {
                    int neighbourCellIndex = parametersPathPlanner.grid.getCellHash(neighbourCellCoordinate);

                    u_int8_t neighborValue = gridValues[neighbourCellIndex];

                    if(neighborValue < stepsToStartCell)
                    {
                        if(neighborValue != stepsToStartCell-1)
                            printf("uh-oh, error1, there's currently %d steps to start, but neighbor value is %d!\n", stepsToStartCell, neighborValue);

                        // prepend our current cell's position to the waypoint list.
                        float3 cellCenter = parametersPathPlanner.grid.getCellCenter(neighbourCellCoordinate);
                        waypoints[neighborValue] = make_float4(cellCenter);

                        printf("retrievePathD(): found by direct step: from cell %d/%d/%d => %d/%d/%d => %d/%d/%d, index %d now at %.2f/%.2f/%.2f\n",
                               currentCellCoordinate.x, currentCellCoordinate.y, currentCellCoordinate.z,
                               travelDirectionDirect.x, travelDirectionDirect.y, travelDirectionDirect.z,
                               neighbourCellCoordinate.x, neighbourCellCoordinate.y, neighbourCellCoordinate.z,
                               neighborValue, cellCenter.x, cellCenter.y, cellCenter.z);

                        // We found a neighbor with a smaller distance. Use it!
                        currentCellCoordinate = neighbourCellCoordinate;

                        // Save this step for the next iteration!
                        lastTravelDirection = travelDirectionDirect;

                        // Escape those 3 for-loops to continue searching from this next cell.
                        foundNextCellTowardsTarget = true;

                        // Update distance to start-position, should be a simple decrement.
                        stepsToStartCell = neighborValue;
                    }
                }
            }

            if(!foundNextCellTowardsTarget)
            {
                // Ok, the direct step didn't work.
                // Define search order. First try to repeat the last step. If that fails, at least try to keep the height.
                int searchOrderX[3];
                if(lastTravelDirection.x == 0)
                {
                    searchOrderX[0] = lastTravelDirection.x;
                    searchOrderX[1] = -1;
                    searchOrderX[2] = +1;
                }
                else
                {
                    searchOrderX[0] = lastTravelDirection.x;
                    searchOrderX[1] = +0;
                    searchOrderX[2] = -lastTravelDirection.x;
                }

                int searchOrderY[3];
                if(lastTravelDirection.y == 0)
                {
                    searchOrderY[0] = lastTravelDirection.y;
                    searchOrderY[1] = -1;
                    searchOrderY[2] = +1;
                }
                else
                {
                    searchOrderY[0] = lastTravelDirection.y;
                    searchOrderY[1] = +0;
                    searchOrderY[2] = -lastTravelDirection.y;
                }

                int searchOrderZ[3];
                if(lastTravelDirection.z == 0)
                {
                    searchOrderZ[0] = lastTravelDirection.z;
                    searchOrderZ[1] = -1;
                    searchOrderZ[2] = +1;}
                else
                {
                    searchOrderZ[0] = lastTravelDirection.z;
                    searchOrderZ[1] = +0;
                    searchOrderZ[2] = -lastTravelDirection.z;
                }

                // now search the neighbors in the given order.
                for(int z=0; z<3 && !foundNextCellTowardsTarget; z++)
                {
                    for(int y=0; y<3 && !foundNextCellTowardsTarget; y++) // check lower paths first
                    {
                        for(int x=0; x<3 && !foundNextCellTowardsTarget; x++)
                        {
                            int3 cellOffset = make_int3(searchOrderX[x], searchOrderY[y], searchOrderZ[z]);
                            int3 neighbourCellCoordinate = currentCellCoordinate + cellOffset;

                            if(parametersPathPlanner.grid.isCellInGrid(neighbourCellCoordinate))
                            {
                                int neighbourCellIndex = parametersPathPlanner.grid.getCellHash(neighbourCellCoordinate);
                                u_int8_t neighborValue = gridValues[neighbourCellIndex];

                                if(neighborValue < stepsToStartCell)
                                {
                                    if(neighborValue != stepsToStartCell-1)
                                        printf("uh-oh, error2, there's currently %d steps to start, but neighbor value is %d!\n", stepsToStartCell, neighborValue);

                                    // Append our current cell's position to the waypoint list.
                                    float3 cellCenter = parametersPathPlanner.grid.getCellCenter(neighbourCellCoordinate);

                                    if(x+y+z == 0)
                                    {
                                        printf("retrievePathD(): found by repeating last step: from cell %d/%d/%d => %d/%d/%d => %d/%d/%d, index %d now at %.2f/%.2f/%.2f\n",
                                               currentCellCoordinate.x, currentCellCoordinate.y, currentCellCoordinate.z,
                                               cellOffset.x, cellOffset.y, cellOffset.z,
                                               neighbourCellCoordinate.x, neighbourCellCoordinate.y, neighbourCellCoordinate.z,
                                               neighborValue, cellCenter.x, cellCenter.y, cellCenter.z);
                                    }
                                    else
                                    {
                                        printf("retrievePathD(): found by searching all neighbors: from cell %d/%d/%d => %d/%d/%d => %d/%d/%d, index %d now at %.2f/%.2f/%.2f\n",
                                               currentCellCoordinate.x, currentCellCoordinate.y, currentCellCoordinate.z,
                                               cellOffset.x, cellOffset.y, cellOffset.z,
                                               neighbourCellCoordinate.x, neighbourCellCoordinate.y, neighbourCellCoordinate.z,
                                               neighborValue, cellCenter.x, cellCenter.y, cellCenter.z);
                                    }

                                    // We found a neighbor with a smaller distance. Use it!
                                    currentCellCoordinate = neighbourCellCoordinate;

                                    lastTravelDirection = cellOffset;

                                    // The w-component doesn't matter here, so set to zero. Later on, the w-component
                                    // will be set to 1 if it turns out that the waypoint is in a now-occupied cell.
                                    waypoints[neighborValue] = make_float4(cellCenter, 0.0);

                                    // Escape those 3 for-loops to continue searching from this next cell.
                                    foundNextCellTowardsTarget = true;

                                    // Update distance to start-position, should be a simple decrement.
                                    stepsToStartCell = neighborValue;
                                }
                            }
                        }
                    }
                }
            }
        }
        while(stepsToStartCell > 1);

        // waypoints[1] was filled above with the cell-center. But we want it to be the start-position, which
        // - although contained in the cell - is probably not exactly its center.
        printf("retrievePathD(): ending, writing start-pos into index 1: %.2f/%.2f/%.2f\n",
               parametersPathPlanner.start.x, parametersPathPlanner.start.y, parametersPathPlanner.start.z);

        waypoints[1] = make_float4(parametersPathPlanner.start);
    }
}


void retrievePath(unsigned char* gridValues, float *waypoints, hipStream_t *stream)
{
    retrievePathD<<< 1, 1, 0, *stream>>>(gridValues, (float4*)waypoints);
    cudaCheckSuccess("retrievePathD");
}

__global__ void testWayPointCellOccupancyD(unsigned char*  gridValues, float4* upcomingWayPoints, unsigned int numberOfWayPoints)
{
    uint waypointIndex = getThreadIndex1D();
    if(waypointIndex >= numberOfWayPoints) return;

    float4 waypoint = upcomingWayPoints[waypointIndex];

    const int3 gridCellCoordinate = parametersPathPlanner.grid.getCellCoordinate(make_float3(waypoint.x, waypoint.y, waypoint.z));
    const int gridCellHash = parametersPathPlanner.grid.getSafeCellHash(gridCellCoordinate);

    if(gridCellHash < 0 || gridCellHash > parametersPathPlanner.grid.getCellCount())
        printf("testWayPointCellOccupancyD(): bug, waypoint %d is supposedly at %.2f/%.2f/%.2f/%.2f in cell hash %d\n",
               waypointIndex, waypoint.x, waypoint.y, waypoint.z, waypoint.w, gridCellHash);

    if(gridValues[gridCellHash] > 253)
    {
        // Waypoints have an information gain (w-component) of either positive for real waypoints
        // or 0 for path-detour waypoints. Flip/Decrement the w-component, so that coliding waypoints
        // can be detected because they have a negative information gain.
        waypoint.w *= -1.0; // detour waypoints are still zero!
        waypoint.w -= 1.0;
        upcomingWayPoints[waypointIndex] = waypoint;
    }
}

void testWayPointCellOccupancy(unsigned char*  gridValues, float* upcomingWayPoints, unsigned int numberOfWayPoints, hipStream_t *stream)
{
    // Start a sufficient number of threads and let the superfuous ones hang out for a while.
    uint numThreads, numBlocks;
    computeExecutionKernelGrid(numberOfWayPoints, 64, numBlocks, numThreads);

    testWayPointCellOccupancyD<<< numBlocks, numThreads, 0, *stream>>>(gridValues, (float4*)upcomingWayPoints, numberOfWayPoints);
    cudaCheckSuccess("testWayPointCellOccupancy");
}
